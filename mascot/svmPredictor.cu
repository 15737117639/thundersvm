#include "hip/hip_runtime.h"
/*
 * @brief: this file contains the definition of svm predictor class
 * Created on: May 24, 2012
 * Author: Zeyi Wen
 * Copyright @DBGroup University of Melbourne
 */

#include "../svm-shared/gpu_global_utility.h"
#include "svmPredictor.h"
#include "../svm-shared/storageManager.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>

/*
 * @brief: read kernel values based on support vectors
 */
void CSVMPredictor::ReadKVbasedOnSV(float_point *pfSVsKernelValues, int *pnSVSampleId, int nNumofSVs, int nNumofTestSamples)
{
	FILE *pFile = fopen(HESSIAN_FILE, "rb");
	float_point *pfSVHessianSubRow = new float_point[nNumofTestSamples];
	float_point *pfHessianFullRow = new float_point[m_pHessianReader->m_nTotalNumofInstance];
	memset(pfSVHessianSubRow, 0, sizeof(float_point) * nNumofTestSamples);

	for(int i = 0; i < nNumofSVs; i++)
	{
		//read part of the Hessian Row

		if(m_pHessianReader->m_nNumofCachedHessianRow > pnSVSampleId[i])
		{
			//if the hessian row is in host memory
			long long nIndexofFirstElement;
			//only one if-statement holds, as testing samples are continuously allocated in RAM
			if(m_pHessianReader->m_nRowStartPos1 != -1)
			{
				nIndexofFirstElement = (long long)pnSVSampleId[i] * m_pHessianReader->m_nTotalNumofInstance +
			  									  m_pHessianReader->m_nRowStartPos1;
			}
			if(m_pHessianReader->m_nRowStartPos2 != -1)
			{
				nIndexofFirstElement = (long long)pnSVSampleId[i] * m_pHessianReader->m_nTotalNumofInstance +
												 m_pHessianReader->m_nRowStartPos2;
			}
			//copy the memory
			memcpy(pfSVHessianSubRow, m_pHessianReader->m_pfHessianRowsInHostMem + nIndexofFirstElement,
					nNumofTestSamples * sizeof(float_point));
		}
		else//the hessian row is in SSD
		{
			int nStartPos;
			if(m_pHessianReader->m_nRowStartPos1 != -1)
			{
				nStartPos = m_pHessianReader->m_nRowStartPos1;
			}
			else if(m_pHessianReader->m_nRowStartPos2 != -1)
			{
				nStartPos = m_pHessianReader->m_nRowStartPos2;
			}
			else
			{
				assert(0);
			}
			m_pHessianReader->ReadRow(pnSVSampleId[i], pfSVHessianSubRow);
			//int nIndexInSSD = pnSVSampleId[i] - m_pHessianOps->m_nNumofCachedHessianRow;
			//m_pHessianOps->ReadHessianFullRow(pFile, nIndexInSSD, 1, pfHessianFullRow);
			//memcpy(pfSVHessianSubRow, pfHessianFullRow + nStartPos, nNumofTestSamples * sizeof(float_point));
		}

		for(int j = 0; j < nNumofTestSamples; j++)
		{
			//store kernel values in a matrix with the form that row is testing samples, column is SVs.
			pfSVsKernelValues[j * (long long)nNumofSVs + i] = pfSVHessianSubRow[j];
		}
	}
	fclose(pFile);
	delete[] pfSVHessianSubRow;
	delete[] pfHessianFullRow;
}

/*
 * @brief: read kernel values based on testing examples
 */
void CSVMPredictor::ReadKVbasedOnTest(float_point *pfSVsKernelValues, int *pnSVSampleId, int nNumofSVs, int nNumofTestSamples)
{
	FILE *pFile = fopen(HESSIAN_FILE, "rb");
	float_point *pfSVHessianSubRow = new float_point[nNumofSVs];
	memset(pfSVHessianSubRow, 0, sizeof(float_point) * nNumofSVs);

	float_point *pfHessianRow = new float_point[m_pHessianReader->m_nTotalNumofInstance];

	int nTestStartId = m_nTestStart;
	assert(nTestStartId >= 0);
	int nTestEndId = nTestStartId + nNumofTestSamples - 1;//include the last sample

	for(int i = nTestStartId; i <= nTestEndId; i++)
	{
		//read part of the Hessian Row
		//if the hessian row is in host memory
		if(m_pHessianReader->m_nNumofCachedHessianRow > i)
		{
			for(int j = 0; j < nNumofSVs; j++)
			{
				pfSVHessianSubRow[j] = m_pHessianReader->m_pfHessianRowsInHostMem[i * (long long)m_pHessianReader->m_nTotalNumofInstance + pnSVSampleId[j]];
			}
		}
		else//the hessian row is in SSD
		{
			m_pHessianReader->ReadRow(i, pfHessianRow);
			for(int j = 0; j < nNumofSVs; j++)
			{
				pfSVHessianSubRow[j] = pfHessianRow[pnSVSampleId[j]];
			}
		}

		for(int j = 0; j < nNumofSVs; j++)
		{
			//store kernel values in a matrix with the form that row is testing samples, column is SVs.
			pfSVsKernelValues[(i - nTestStartId) * (long long)nNumofSVs + j] = pfSVHessianSubRow[j];
		}
	}
	if(pFile != NULL)
		fclose(pFile);
	delete[] pfSVHessianSubRow;
	delete[] pfHessianRow;
}
/*
 * @brief: predict class labels
 */
float_point* CSVMPredictor::Predict(svm_model *pModel, int *pnTestSampleId, const int &nNumofTestSamples)
{
	float_point *pfReturn = NULL;
	if(pModel == NULL)
	{
		cerr << "error in Predict function: invalid input params" << endl;
		return pfReturn;
	}

	//get infomation from SVM model
	int nNumofSVs = pModel->nSV[0] + pModel->nSV[1];
	float_point fBias = *(pModel->rho);
	float_point **pyfSVsYiAlpha = pModel->sv_coef;
	float_point *pfSVsYiAlpha = pyfSVsYiAlpha[0];
	int *pnSVsLabel = pModel->label;
	int *pnSVSampleId = pModel->pnIndexofSV;

	//store sub Hessian Matrix
	float_point *pfSVsKernelValues = new float_point[nNumofTestSamples * nNumofSVs];
	memset(pfSVsKernelValues, 0, sizeof(float_point) * nNumofTestSamples * nNumofSVs);

	float_point *pfYiAlphaofSVs;

	//get Hessian rows of support vectors
	m_pHessianReader->AllocateBuffer(1);
	if(nNumofSVs >= nNumofTestSamples)
	{
		m_pHessianReader->SetInvolveData(-1, -1, 0, m_pHessianReader->m_nTotalNumofInstance - 1);
		ReadKVbasedOnTest(pfSVsKernelValues, pnSVSampleId, nNumofSVs, nNumofTestSamples);
	}
	else
	{
		m_pHessianReader->SetInvolveData(-1, -1, pnTestSampleId[0], pnTestSampleId[nNumofTestSamples - 1]);
		ReadKVbasedOnSV(pfSVsKernelValues, pnSVSampleId, nNumofSVs, nNumofTestSamples);
	}
	m_pHessianReader->ReleaseBuffer();

	/*compute y_i*alpha_i*K(i, z) by GPU, where i is id of support vector.
	 * pfDevSVYiAlphaHessian stores in the order of T1 sv1 sv2 ... T2 sv1 sv2 ... T3 sv1 sv2 ...
	 */
	float_point *pfDevSVYiAlphaHessian;
	float_point *pfDevSVsYiAlpha;
	int *pnDevSVsLabel;

	//if the memory is not enough for the storage when classifying all testing samples at once, divide it into multiple parts

	StorageManager *manager = StorageManager::getManager();
	int nMaxNumofFloatPoint = manager->GetFreeGPUMem();
	int nNumofPart = Ceil(nNumofSVs * nNumofTestSamples, nMaxNumofFloatPoint);

//	cout << "cache size is: " << nMaxNumofFloatPoint << " v.s.. " << nNumofSVs * nNumofTestSamples << endl;
//	cout << "perform classification in " << nNumofPart << " time(s)" << endl;

	//allocate memory for storing classification result
	float_point *pfClassificaitonResult = new float_point[nNumofTestSamples];
	//initialise the size of each part
	int *pSizeofPart = new int[nNumofPart];
	int nAverageSize = nNumofTestSamples / nNumofPart;
	for(int i = 0; i < nNumofPart; i++)
	{
		if(i != nNumofPart - 1)
		{
			pSizeofPart[i] = nAverageSize;
		}
		else
		{
			pSizeofPart[i] = nNumofTestSamples - nAverageSize * i;
		}
	}

	//perform classification for each part
	for(int i = 0; i < nNumofPart; i++)
	{
	checkCudaErrors(hipMalloc((void**)&pfDevSVYiAlphaHessian, sizeof(float_point) * nNumofSVs * pSizeofPart[i]));
	checkCudaErrors(hipMalloc((void**)&pfDevSVsYiAlpha, sizeof(float_point) * nNumofSVs));
	checkCudaErrors(hipMalloc((void**)&pnDevSVsLabel, sizeof(int) * nNumofSVs));

	checkCudaErrors(hipMemset(pfDevSVYiAlphaHessian, 0, sizeof(float_point) * nNumofSVs * pSizeofPart[i]));
	checkCudaErrors(hipMemset(pfDevSVsYiAlpha, 0, sizeof(float_point) * nNumofSVs));
	checkCudaErrors(hipMemset(pnDevSVsLabel, 0, sizeof(int) * nNumofSVs));

	checkCudaErrors(hipMemcpy(pfDevSVYiAlphaHessian, pfSVsKernelValues + i * nAverageSize * nNumofSVs,
				  	  	  	   sizeof(float_point) * nNumofSVs * pSizeofPart[i], hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pfDevSVsYiAlpha, pfSVsYiAlpha, sizeof(float_point) * nNumofSVs, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pnDevSVsLabel, pnSVsLabel, sizeof(int) * nNumofSVs, hipMemcpyHostToDevice));

	//compute y_i*alpha_i*K(i, z)
	int nVecMatxMulGridDimY = pSizeofPart[i];
	int nVecMatxMulGridDimX = Ceil(nNumofSVs, BLOCK_SIZE);
	dim3 vecMatxMulGridDim(nVecMatxMulGridDimX, nVecMatxMulGridDimY);
	VectorMatrixMul<<<vecMatxMulGridDim, BLOCK_SIZE>>>(pfDevSVsYiAlpha, pfDevSVYiAlphaHessian, pSizeofPart[i], nNumofSVs);

	//perform classification
	ComputeClassLabel(pSizeofPart[i], pfDevSVYiAlphaHessian,
					  nNumofSVs, fBias, pfClassificaitonResult + i * nAverageSize);

	if(pfClassificaitonResult == NULL)
	{
		cerr << "error in ComputeClassLabel" << endl;
		return pfReturn;
	}


	//free memory
	checkCudaErrors(hipFree(pfDevSVYiAlphaHessian));
	pfDevSVYiAlphaHessian = NULL;
	checkCudaErrors(hipFree(pfDevSVsYiAlpha));
	checkCudaErrors(hipFree(pnDevSVsLabel));
	}

	delete[] pfSVsKernelValues;

	pfReturn = pfClassificaitonResult;
	return pfReturn;
}

/*
float_point* CSVMPredictor::Predict(svm_model *pModel, svm_node **pInstance, const int &numInstance)
{
	float_point *pfReturn = NULL;
	if(pModel == NULL)
	{
		cerr << "error in Predict function: invalid input params" << endl;
		return pfReturn;
	}

	//get infomation from SVM model
	int nNumofSVs = pModel->nSV[0] + pModel->nSV[1];
	float_point fBias = *(pModel->rho);
	float_point **pyfSVsYiAlpha = pModel->sv_coef;
	float_point *pfSVsYiAlpha = pyfSVsYiAlpha[0];
	int *pnSVsLabel = pModel->label;
	int *pnSVSampleId = pModel->pnIndexofSV;

	//store sub Hessian Matrix
	float_point *pfSVsKernelValues = new float_point[numInstance * nNumofSVs];
	memset(pfSVsKernelValues, 0, sizeof(float_point) * numInstance * nNumofSVs);

	float_point *pfYiAlphaofSVs;

	//get Hessian rows of support vectors
	m_pHessianReader->AllocateBuffer(1);
	if(nNumofSVs >= numInstance)
	{
		m_pHessianReader->SetInvolveData(-1, -1, 0, m_pHessianReader->m_nTotalNumofInstance - 1);
		ReadKVbasedOnTest(pfSVsKernelValues, pnSVSampleId, nNumofSVs, numInstance);
	}
	else
	{
		m_pHessianReader->SetInvolveData(-1, -1, pnTestSampleId[0], pnTestSampleId[nNumofTestSamples - 1]);
		ReadKVbasedOnSV(pfSVsKernelValues, pnSVSampleId, nNumofSVs, numInstance);
	}
	m_pHessianReader->ReleaseBuffer();

	/*compute y_i*alpha_i*K(i, z) by GPU, where i is id of support vector.
	 * pfDevSVYiAlphaHessian stores in the order of T1 sv1 sv2 ... T2 sv1 sv2 ... T3 sv1 sv2 ...
	 */
/*	float_point *pfDevSVYiAlphaHessian;
	float_point *pfDevSVsYiAlpha;
	int *pnDevSVsLabel;

	//if the memory is not enough for the storage when classifying all testing samples at once, divide it into multiple parts

	StorageManager *manager = StorageManager::getManager();
	int nMaxNumofFloatPoint = manager->GetFreeGPUMem();
	int nNumofPart = Ceil(nNumofSVs * numInstance, nMaxNumofFloatPoint);

//	cout << "cache size is: " << nMaxNumofFloatPoint << " v.s.. " << nNumofSVs * nNumofTestSamples << endl;
//	cout << "perform classification in " << nNumofPart << " time(s)" << endl;

	//allocate memory for storing classification result
	float_point *pfClassificaitonResult = new float_point[numInstance];
	//initialise the size of each part
	int *pSizeofPart = new int[nNumofPart];
	int nAverageSize = numInstance / nNumofPart;
	for(int i = 0; i < nNumofPart; i++)
	{
		if(i != nNumofPart - 1)
		{
			pSizeofPart[i] = nAverageSize;
		}
		else
		{
			pSizeofPart[i] = numInstance - nAverageSize * i;
		}
	}

	//perform classification for each part
	for(int i = 0; i < nNumofPart; i++)
	{
	checkCudaErrors(hipMalloc((void**)&pfDevSVYiAlphaHessian, sizeof(float_point) * nNumofSVs * pSizeofPart[i]));
	checkCudaErrors(hipMalloc((void**)&pfDevSVsYiAlpha, sizeof(float_point) * nNumofSVs));
	checkCudaErrors(hipMalloc((void**)&pnDevSVsLabel, sizeof(int) * nNumofSVs));

	checkCudaErrors(hipMemset(pfDevSVYiAlphaHessian, 0, sizeof(float_point) * nNumofSVs * pSizeofPart[i]));
	checkCudaErrors(hipMemset(pfDevSVsYiAlpha, 0, sizeof(float_point) * nNumofSVs));
	checkCudaErrors(hipMemset(pnDevSVsLabel, 0, sizeof(int) * nNumofSVs));

	checkCudaErrors(hipMemcpy(pfDevSVYiAlphaHessian, pfSVsKernelValues + i * nAverageSize * nNumofSVs,
				  	  	  	   sizeof(float_point) * nNumofSVs * pSizeofPart[i], hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pfDevSVsYiAlpha, pfSVsYiAlpha, sizeof(float_point) * nNumofSVs, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(pnDevSVsLabel, pnSVsLabel, sizeof(int) * nNumofSVs, hipMemcpyHostToDevice));

	//compute y_i*alpha_i*K(i, z)
	int nVecMatxMulGridDimY = pSizeofPart[i];
	int nVecMatxMulGridDimX = Ceil(nNumofSVs, BLOCK_SIZE);
	dim3 vecMatxMulGridDim(nVecMatxMulGridDimX, nVecMatxMulGridDimY);
	VectorMatrixMul<<<vecMatxMulGridDim, BLOCK_SIZE>>>(pfDevSVsYiAlpha, pfDevSVYiAlphaHessian, pSizeofPart[i], nNumofSVs);

	//perform classification
	ComputeClassLabel(pSizeofPart[i], pfDevSVYiAlphaHessian,
					  nNumofSVs, fBias, pfClassificaitonResult + i * nAverageSize);

	if(pfClassificaitonResult == NULL)
	{
		cerr << "error in ComputeClassLabel" << endl;
		return pfReturn;
	}


	//free memory
	checkCudaErrors(hipFree(pfDevSVYiAlphaHessian));
	pfDevSVYiAlphaHessian = NULL;
	checkCudaErrors(hipFree(pfDevSVsYiAlpha));
	checkCudaErrors(hipFree(pnDevSVsLabel));
	}

	delete[] pfSVsKernelValues;

	pfReturn = pfClassificaitonResult;
	return pfReturn;
}
*/

/*
 * @brief: compute/predict the labels of testing samples
 * @output: a set of class labels, associated to testing samples
 */
float_point* CSVMPredictor::ComputeClassLabel(int nNumofTestSamples,
									  float_point *pfDevSVYiAlphaHessian, const int &nNumofSVs,
									  float_point fBias, float_point *pfFinalResult)
{
	float_point *pfReturn = NULL;
	if(nNumofTestSamples <= 0 ||
	   pfDevSVYiAlphaHessian == NULL ||
	   nNumofSVs <= 0)
	{
		cerr << "error in ComputeClassLabel: invalid input params" << endl;
		return pfReturn;
	}

	//compute the size of current processing testing samples
/*	size_t nFreeMemory,nTotalMemory;
	hipMemGetInfo(&nFreeMemory,&nTotalMemory);
*/	int nMaxSizeofProcessingSample = ((CACHE_SIZE) * 1024 * 1024 * 4 / (sizeof(float_point) * nNumofSVs));

	//reduce by half
	nMaxSizeofProcessingSample = nMaxSizeofProcessingSample / 2;

	//if the number of samples in small
	if(nMaxSizeofProcessingSample > nNumofTestSamples)
	{
		nMaxSizeofProcessingSample = nNumofTestSamples;
	}
	//compute grid size, and block size for partial sum
	int nPartialGridDimX = Ceil(nNumofSVs, BLOCK_SIZE);
	int nPartialGridDimY = nMaxSizeofProcessingSample;
	dim3 dimPartialSumGrid(nPartialGridDimX, nPartialGridDimY);
	dim3 dimPartialSumBlock(BLOCK_SIZE);

	//compute grid size, and block size for global sum and class label computing
	int nGlobalGridDimX = 1;
	int nGlobalGridDimY = nMaxSizeofProcessingSample;
	dim3 dimGlobalSumGrid(nGlobalGridDimX, nGlobalGridDimY); //can use 1D grid
	dim3 dimGlobalSumBlock(nPartialGridDimX);

	//memory for computing partial sum by GPU
	float_point* pfDevPartialSum;
//	cout << "dimx=" << nPartialGridDimX << "; dimy=" << nPartialGridDimY << endl;
	checkCudaErrors(hipMalloc((void**)&pfDevPartialSum, sizeof(float_point) * nPartialGridDimX * nPartialGridDimY));
	checkCudaErrors(hipMemset(pfDevPartialSum, 0, sizeof(float_point) * nPartialGridDimX * nPartialGridDimY));

	//memory for computing global sum by GPU
	float_point *pfDevClassificationResult;
	checkCudaErrors(hipMalloc((void**)&pfDevClassificationResult, sizeof(float_point) * nGlobalGridDimY));
	checkCudaErrors(hipMemset(pfDevClassificationResult, 0, sizeof(float_point) * nGlobalGridDimY));

	//reduce step size of partial sum, and global sum
	int nPartialReduceStepSize = 0;
	nPartialReduceStepSize = (int)pow(2, (ceil(log2((float)BLOCK_SIZE))-1));
	int nGlobalReduceStepSize = 0;
	nGlobalReduceStepSize = (int) pow(2, ceil(log2((float) nPartialGridDimX)) - 1);

	for(int nStartPosofTestSample = 0; nStartPosofTestSample < nNumofTestSamples; nStartPosofTestSample += nMaxSizeofProcessingSample)
	{
		if(nStartPosofTestSample + nMaxSizeofProcessingSample > nNumofTestSamples)
		{
			//the last part of the testing samples
			nMaxSizeofProcessingSample = nNumofTestSamples - nStartPosofTestSample;
			nPartialGridDimY = nMaxSizeofProcessingSample;
			dimPartialSumGrid = dim3(nPartialGridDimX, nPartialGridDimY);
			nGlobalGridDimY = nMaxSizeofProcessingSample;
			dimGlobalSumGrid = dim3(nGlobalGridDimX, nGlobalGridDimY);

			checkCudaErrors(hipFree(pfDevPartialSum));
			checkCudaErrors(hipMalloc((void**)&pfDevPartialSum, sizeof(float_point) * nPartialGridDimX * nPartialGridDimY));
			checkCudaErrors(hipMemset(pfDevPartialSum, 0, sizeof(float_point) * nPartialGridDimX * nPartialGridDimY));

			checkCudaErrors(hipFree(pfDevClassificationResult));
			checkCudaErrors(hipMalloc((void**)&pfDevClassificationResult, sizeof(float_point) * nGlobalGridDimY));
			checkCudaErrors(hipMemset(pfDevClassificationResult, 0, sizeof(float_point) * nGlobalGridDimY));
		}
		/********* compute partial sum **********/
		ComputeKernelPartialSum<<<dimPartialSumGrid, dimPartialSumBlock, BLOCK_SIZE * sizeof(float_point)>>>
							   (pfDevSVYiAlphaHessian, nNumofSVs, pfDevPartialSum,
								nPartialReduceStepSize);
		hipError_t error = hipDeviceSynchronize();
		if(error != hipSuccess)
		{
			cerr << "cuda error in ComputeClassLabel: failed at ComputePartialSum: " << hipGetErrorString(error) << endl;
			return pfReturn;
		}

		/********** compute global sum and class label *********/
		//compute global sum
		ComputeKernelGlobalSum<<<dimGlobalSumGrid, dimGlobalSumBlock, nPartialGridDimX * sizeof(float_point)>>>
							  (pfDevClassificationResult, fBias,
							   pfDevPartialSum, nGlobalReduceStepSize);
		hipDeviceSynchronize();

		error = hipGetLastError();
		if(error != hipSuccess)
		{
			cerr << "cuda error in ComputeClassLabel: failed at ComputeGlobalSum: " << hipGetErrorString(error) << endl;
			return pfReturn;
		}

		//copy classification result back
		checkCudaErrors(hipMemcpy(pfFinalResult + nStartPosofTestSample, pfDevClassificationResult,
								 nMaxSizeofProcessingSample * sizeof(float_point), hipMemcpyDeviceToHost));
	}

	checkCudaErrors(hipFree(pfDevPartialSum));
	checkCudaErrors(hipFree(pfDevClassificationResult));

	pfReturn = pfFinalResult;
	return pfReturn;
}

/*
 * @brief: set data involved in prediction
 */
bool CSVMPredictor::SetInvolvePredictionData(int nStart1, int nEnd1)
{
	bool bReturn = false;
	m_nTestStart = nStart1;
	bReturn = m_pHessianReader->SetInvolveData(-1, -1, 0, m_pHessianReader->m_nTotalNumofInstance - 1);

	return bReturn;
}
