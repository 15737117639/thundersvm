#include "hip/hip_runtime.h"
/*
   * @author: created by ss on 16-11-2.
   * @brief: multi-class svm training, prediction, svm with probability output
   *
*/

#include "svmModel.h"

#include "svmPredictor.h"
#include "../svm-shared/HessianIO/deviceHessian.h"
#include "../svm-shared/storageManager.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include "trainingFunction.h"

//todo move these kernel functions to a proper file
__global__ void
rbfKernel(const float_point *samples, int numOfSamples, const float_point *supportVectors, int numOfSVs,
          int numOfFeatures,
          float_point *kernelValues, float_point gamma,
          const float_point *coef) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int sampleId = idx / numOfSVs;
    int SVId = idx % numOfSVs;
    if (sampleId < numOfSamples) {
        const float_point *sample = samples + sampleId * numOfFeatures;
        const float_point *supportVector = supportVectors + SVId * numOfFeatures;
        float_point sum = 0;
        for (int i = 0; i < numOfFeatures; ++i) {
            float_point d = sample[i] - supportVector[i];
            sum += d * d;
        }
        kernelValues[idx] = coef[SVId] * exp(-gamma * sum);
    }
};

__global__ void sumKernelValues(const float *kernelValues, int numOfSamples, int numOfSVs, int cnr2,
                                const int *start, const int *count,
                                const float *bias, float_point *decValues) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int sampleId = idx / cnr2;
    int modelId = idx % cnr2;
    if (sampleId < numOfSamples) {
        float_point sum = 0;
        const float_point *kernelValue = kernelValues + sampleId * numOfSVs + start[modelId];
        for (int i = 0; i < count[modelId]; ++i) {
            sum += kernelValue[i];
        }
        decValues[idx] = sum - bias[modelId];
    }
}

SvmModel::~SvmModel() {
    checkCudaErrors(hipFree(devSVs));
    checkCudaErrors(hipFree(devCoef));
    checkCudaErrors(hipFree(devStart));
    checkCudaErrors(hipFree(devCount));
    checkCudaErrors(hipFree(devProbA));
    checkCudaErrors(hipFree(devProbB));
    checkCudaErrors(hipFree(devRho));
}

unsigned int SvmModel::getK(int i, int j) const {
    return ((nrClass - 1) + (nrClass - i)) * i / 2 + j - i - 1;
}

void SvmModel::fit(const SvmProblem &problem, const SVMParam &param) {
    //reset model to fit a new SvmProblem
    nrClass = problem.getNumOfClasses();
    cnr2 = (nrClass) * (nrClass - 1) / 2;
    numOfFeatures = problem.v_vSamples.front().size();
    numOfSVs = 0;
    coef.clear();
    rho.clear();
    probA.clear();
    probB.clear();
    supportVectors.clear();
    label.clear();
    start.clear();
    count.clear();
    probability = false;

    coef.resize(cnr2);
    rho.resize(cnr2);
    probA.resize(cnr2);
    probB.resize(cnr2);
    supportVectors.resize(cnr2);

    this->param = param;
    label = problem.label;

    //train nrClass*(nrClass-1)/2 binary models
    int k = 0;
    for (int i = 0; i < nrClass; ++i) {
        for (int j = i + 1; j < nrClass; ++j) {
            SvmProblem subProblem = problem.getSubProblem(i, j);
            printf("training classifier with label %d and %d\n", i, j);
            if (param.probability) {
                SVMParam probParam = param;
                probParam.probability = 0;
                probParam.C = 1.0;
                SvmModel model;
                model.fit(subProblem, probParam);
                vector<vector<float_point> > decValues;
                //todo predict with cross validation
                model.predictValues(subProblem.v_vSamples, decValues);
                for (int l = 1; l < subProblem.v_vSamples.size(); ++l) {
                    decValues[0].push_back(decValues[l][0]);
                }
                sigmoidTrain(decValues.front().data(), subProblem.getNumOfSamples(), subProblem.v_nLabels, probA[k],
                             probB[k]);
                probability = true;
            }
            svm_model binaryModel = trainBinarySVM(subProblem, param);
            addBinaryModel(subProblem, binaryModel, i, j);
            k++;
        }
    }
    int _start = 0;
    for (int i = 0; i < cnr2; ++i) {
        start.push_back(_start);
        count.push_back(supportVectors[i].size());
        _start += count[i];
    }
    transferToDevice();
}

void SvmModel::transferToDevice() {
    int svLength = numOfFeatures;
    checkCudaErrors(hipMalloc((void **) &devSVs, sizeof(float_point) * numOfSVs * svLength));
    checkCudaErrors(hipMalloc((void **) &devCoef, sizeof(float_point) * numOfSVs));
    checkCudaErrors(hipMalloc((void **) &devStart, sizeof(float_point) * cnr2));
    checkCudaErrors(hipMalloc((void **) &devCount, sizeof(float_point) * cnr2));
    checkCudaErrors(hipMalloc((void **) &devProbA, sizeof(float_point) * cnr2));
    checkCudaErrors(hipMalloc((void **) &devProbB, sizeof(float_point) * cnr2));
    checkCudaErrors(hipMalloc((void **) &devRho, sizeof(float_point) * cnr2));
    for (int i = 0; i < cnr2; ++i) {
        float_point *sv4BinaryModel = new float_point[supportVectors[i].size() * svLength];
        for (int j = 0; j < supportVectors[i].size(); ++j) {
            memcpy(sv4BinaryModel + j * svLength, supportVectors[i][j].data(), sizeof(float_point) * svLength);
        }
        checkCudaErrors(hipMemcpy(devSVs + start[i] * svLength, sv4BinaryModel,
                                   sizeof(float_point) * count[i] * svLength, hipMemcpyHostToDevice));
        delete[] sv4BinaryModel;
        checkCudaErrors(hipMemcpy(devCoef + start[i], coef[i].data(), sizeof(float_point) * count[i],
                                   hipMemcpyHostToDevice));
    }
    checkCudaErrors(hipMemcpy(devProbA, probA.data(), sizeof(float_point) * cnr2, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(devProbB, probB.data(), sizeof(float_point) * cnr2, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(devStart, start.data(), sizeof(int) * cnr2, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(devCount, count.data(), sizeof(int) * cnr2, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(devRho, rho.data(), sizeof(float_point) * cnr2, hipMemcpyHostToDevice));
}

void SvmModel::sigmoidTrain(const float_point *decValues, const int l, const vector<int> &labels, float_point &A,
                            float_point &B) {
    double prior1 = 0, prior0 = 0;
    int i;

    for (i = 0; i < l; i++)
        if (labels[i] > 0)
            prior1 += 1;
        else
            prior0 += 1;

    int max_iter = 100;    // Maximal number of iterations
    double min_step = 1e-10;    // Minimal step taken in line search
    double sigma = 1e-12;    // For numerically strict PD of Hessian
    double eps = 1e-5;
    double hiTarget = (prior1 + 1.0) / (prior1 + 2.0);
    double loTarget = 1 / (prior0 + 2.0);
    double *t = (double *) malloc(sizeof(double) * l);
    double fApB, p, q, h11, h22, h21, g1, g2, det, dA, dB, gd, stepsize;
    double newA, newB, newf, d1, d2;
    int iter;

    // Initial Point and Initial Fun Value
    A = 0.0;
    B = log((prior0 + 1.0) / (prior1 + 1.0));
    double fval = 0.0;

    for (i = 0; i < l; i++) {
        if (labels[i] > 0)
            t[i] = hiTarget;
        else
            t[i] = loTarget;
        fApB = decValues[i] * A + B;
        if (fApB >= 0)
            fval += t[i] * fApB + log(1 + exp(-fApB));
        else
            fval += (t[i] - 1) * fApB + log(1 + exp(fApB));
    }
    for (iter = 0; iter < max_iter; iter++) {
        // Update Gradient and Hessian (use H' = H + sigma I)
        h11 = sigma; // numerically ensures strict PD
        h22 = sigma;
        h21 = 0.0;
        g1 = 0.0;
        g2 = 0.0;
        for (i = 0; i < l; i++) {
            fApB = decValues[i] * A + B;
            if (fApB >= 0) {
                p = exp(-fApB) / (1.0 + exp(-fApB));
                q = 1.0 / (1.0 + exp(-fApB));
            } else {
                p = 1.0 / (1.0 + exp(fApB));
                q = exp(fApB) / (1.0 + exp(fApB));
            }
            d2 = p * q;
            h11 += decValues[i] * decValues[i] * d2;
            h22 += d2;
            h21 += decValues[i] * d2;
            d1 = t[i] - p;
            g1 += decValues[i] * d1;
            g2 += d1;
        }

        // Stopping Criteria
        if (fabs(g1) < eps && fabs(g2) < eps)
            break;

        // Finding Newton direction: -inv(H') * g
        det = h11 * h22 - h21 * h21;
        dA = -(h22 * g1 - h21 * g2) / det;
        dB = -(-h21 * g1 + h11 * g2) / det;
        gd = g1 * dA + g2 * dB;

        stepsize = 1;        // Line Search
        while (stepsize >= min_step) {
            newA = A + stepsize * dA;
            newB = B + stepsize * dB;

            // New function value
            newf = 0.0;
            for (i = 0; i < l; i++) {
                fApB = decValues[i] * newA + newB;
                if (fApB >= 0)
                    newf += t[i] * fApB + log(1 + exp(-fApB));
                else
                    newf += (t[i] - 1) * fApB + log(1 + exp(fApB));
            }
            // Check sufficient decrease
            if (newf < fval + 0.0001 * stepsize * gd) {
                A = newA;
                B = newB;
                fval = newf;
                break;
            } else
                stepsize = stepsize / 2.0;
        }

        if (stepsize < min_step) {
            printf("Line search fails in two-class probability estimates\n");
            break;
        }
    }

    if (iter >= max_iter)
        printf(
                "Reaching maximal iterations in two-class probability estimates\n");
    free(t);
}

void SvmModel::addBinaryModel(const SvmProblem &problem, const svm_model &bModel, int i, int j) {
    unsigned int k = getK(i, j);
    supportVectors[k].resize(bModel.nSV[0] + bModel.nSV[1]);
    for (int l = 0; l < bModel.nSV[0] + bModel.nSV[1]; ++l) {
        coef[k].push_back(bModel.sv_coef[0][l]);
        supportVectors[k][l] = problem.v_vSamples[bModel.pnIndexofSV[l]];
    }
    rho[k] = bModel.rho[0];
    numOfSVs += bModel.nSV[0] + bModel.nSV[1];
}

void
SvmModel::predictValues(const vector<vector<float_point> > &v_vSamples,
                        vector<vector<float_point> > &decisionValues) const {
    //copy samples to device
    float_point *devSamples;
    checkCudaErrors(hipMalloc((void **) &devSamples, sizeof(float_point) * v_vSamples.size() * numOfFeatures));
    for (int i = 0; i < v_vSamples.size(); ++i) {
        checkCudaErrors(hipMemcpy(devSamples + i * numOfFeatures, v_vSamples[i].data(),
                                   sizeof(float_point) * numOfFeatures, hipMemcpyHostToDevice));
    }


    float_point *devKernelValues;
    checkCudaErrors(hipMalloc((void **) &devKernelValues,
                               sizeof(float_point) * v_vSamples.size() * numOfSVs));
    int numOfBlock = Ceil(v_vSamples.size() * numOfSVs, BLOCK_SIZE);
    rbfKernel << < numOfBlock, BLOCK_SIZE >> > (devSamples, v_vSamples.size(),
            devSVs, numOfSVs, numOfFeatures, devKernelValues, param.gamma, devCoef);
    numOfBlock = Ceil(v_vSamples.size() * cnr2, BLOCK_SIZE);
    float_point *devDecisionValues;
    checkCudaErrors(hipMalloc((void **) &devDecisionValues, sizeof(float_point) * v_vSamples.size() * cnr2));
    sumKernelValues << < numOfBlock, BLOCK_SIZE >> > (devKernelValues, v_vSamples.size(),
            numOfSVs, cnr2, devStart, devCount, devRho, devDecisionValues);
    float_point *tempDecValues = new float_point[v_vSamples.size() * cnr2];
    checkCudaErrors(hipMemcpy(tempDecValues, devDecisionValues,
                               sizeof(float_point) * v_vSamples.size() * cnr2, hipMemcpyDeviceToHost));
    decisionValues = vector<vector<float_point> >(v_vSamples.size(), vector<float_point>(cnr2));
    for (int i = 0; i < decisionValues.size(); ++i) {
        memcpy(decisionValues[i].data(), tempDecValues + i * cnr2, sizeof(float_point) * cnr2);
    }
    delete[] tempDecValues;
    checkCudaErrors(hipFree(devSamples));
    checkCudaErrors(hipFree(devDecisionValues));
    checkCudaErrors(hipFree(devKernelValues));
}

vector<int> SvmModel::predict(const vector<vector<float_point> > &v_vSamples, bool probability) const {
    vector<int> labels;
    if (!probability) {
        vector<vector<float_point> > decisionValues;
        predictValues(v_vSamples, decisionValues);
        for (int l = 0; l < v_vSamples.size(); ++l) {
            vector<int> votes(nrClass, 0);
            int k = 0;
            for (int i = 0; i < nrClass; ++i) {
                for (int j = i + 1; j < nrClass; ++j) {
                    if (decisionValues[l][k++] > 0)
                        votes[i]++;
                    else
                        votes[j]++;
                }
            }
            int maxVoteClass = 0;
            for (int i = 0; i < nrClass; ++i) {
                if (votes[i] > votes[maxVoteClass])
                    maxVoteClass = i;
            }
            labels.push_back(this->label[maxVoteClass]);
        }
    } else {
        printf("predict with probability\n");
        assert(this->probability);
        vector<vector<float_point> > prob = predictProbability(v_vSamples);
        // todo select max using GPU
        for (int i = 0; i < v_vSamples.size(); ++i) {
            int maxProbClass = 0;
            for (int j = 0; j < nrClass; ++j) {
                if (prob[i][j] > prob[i][maxProbClass])
                    maxProbClass = j;
            }
            labels.push_back(this->label[maxProbClass]);
        }
    }
    return labels;
}

float_point SvmModel::sigmoidPredict(float_point decValue, float_point A, float_point B) const {
    double fApB = decValue * A + B;
    // 1-p used later; avoid catastrophic cancellation
    if (fApB >= 0)
        return exp(-fApB) / (1.0 + exp(-fApB));
    else
        return 1.0 / (1 + exp(fApB));
}

void SvmModel::multiClassProbability(const vector<vector<float_point> > &r, vector<float_point> &p) const {
    int t, j;
    int iter = 0, max_iter = max(100, nrClass);
    double **Q = (double **) malloc(sizeof(double *) * nrClass);
    double *Qp = (double *) malloc(sizeof(double) * nrClass);
    double pQp, eps = 0.005 / nrClass;

    for (t = 0; t < nrClass; t++) {
        p[t] = 1.0 / nrClass;  // Valid if k = 1
        Q[t] = (double *) malloc(sizeof(double) * nrClass);
        Q[t][t] = 0;
        for (j = 0; j < t; j++) {
            Q[t][t] += r[j][t] * r[j][t];
            Q[t][j] = Q[j][t];
        }
        for (j = t + 1; j < nrClass; j++) {
            Q[t][t] += r[j][t] * r[j][t];
            Q[t][j] = -r[j][t] * r[t][j];
        }
    }
    for (iter = 0; iter < max_iter; iter++) {
        // stopping condition, recalculate QP,pQP for numerical accuracy
        pQp = 0;
        for (t = 0; t < nrClass; t++) {
            Qp[t] = 0;
            for (j = 0; j < nrClass; j++)
                Qp[t] += Q[t][j] * p[j];
            pQp += p[t] * Qp[t];
        }
        double max_error = 0;
        for (t = 0; t < nrClass; t++) {
            double error = fabs(Qp[t] - pQp);
            if (error > max_error)
                max_error = error;
        }
        if (max_error < eps)
            break;

        for (t = 0; t < nrClass; t++) {
            double diff = (-Qp[t] + pQp) / Q[t][t];
            p[t] += diff;
            pQp = (pQp + diff * (diff * Q[t][t] + 2 * Qp[t])) / (1 + diff)
                  / (1 + diff);
            for (j = 0; j < nrClass; j++) {
                Qp[j] = (Qp[j] + diff * Q[t][j]) / (1 + diff);
                p[j] /= (1 + diff);
            }
        }
    }
    if (iter >= max_iter)
        printf("Exceeds max_iter in multiclass_prob\n");
    for (t = 0; t < nrClass; t++)
        free(Q[t]);
    free(Q);
    free(Qp);
}

vector<vector<float_point> > SvmModel::predictProbability(const vector<vector<float_point> > &v_vSamples) const {
    vector<vector<float_point> > result;
    vector<vector<float_point> > decValues;
    predictValues(v_vSamples, decValues);
    for (int l = 0; l < v_vSamples.size(); ++l) {
        vector<vector<float_point> > r(nrClass, vector<float_point>(nrClass));
        double min_prob = 1e-7;
        int k = 0;
        for (int i = 0; i < nrClass; i++)
            for (int j = i + 1; j < nrClass; j++) {
                r[i][j] = min(
                        max(sigmoidPredict(decValues[l][k], probA[k], probB[k]), min_prob), 1 - min_prob);
                r[j][i] = 1 - r[i][j];
                k++;
            }
        vector<float_point> p(nrClass);
        multiClassProbability(r, p);
        result.push_back(p);
    }
    return result;
}

bool SvmModel::isProbability() const {
    return probability;
}

