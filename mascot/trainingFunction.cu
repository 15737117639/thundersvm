#include "hip/hip_runtime.h"
/*
 * testTrainer.cpp
 *
 *  Created on: 31/10/2013
 *      Author: Zeyi
 */

#include "trainingFunction.h"

#include<iostream>
#include<cassert>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <sys/time.h>
#include <sys/sysinfo.h>

#include "../svm-shared/gpu_global_utility.h"
#include "../svm-shared/constant.h"
#include "../svm-shared/HessianIO/baseHessian.h"
#include "../svm-shared/HessianIO/seqAccessor.h"
#include "../svm-shared/kernelCalculater/kernelCalculater.h"
#include "../svm-shared/svmTrainer.h"
#include "../svm-shared/smoSolver.h"
#include "../svm-shared/Cache/cache.h"
#include "DataIOOps/DataIO.h"
#include "DataIOOps/BaseLibsvmReader.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>

#include "svmProblem.h"
#include "../svm-shared/HessianIO/deviceHessianOnFly.h"

extern long nTimeOfLoop;
extern long lGetHessianRowTime;
extern long readRowTime;
extern long lGetHessianRowCounter;
extern long cacheMissCount;
extern long cacheMissMemcpyTime;
using std::cout;
using std::endl;

void trainSVM(SVMParam &param, string strTrainingFileName, int nNumofFeature, SvmModel &model) {

    clock_t start, end;
    vector<vector<svm_node> > v_v_DocVector;
    vector<int> v_nLabel;

    CDataIOOps rawDataRead;
    int nNumofInstance = 0;     //not used
    long long nNumofValue = 0;  //not used
    BaseLibSVMReader::GetDataInfo(strTrainingFileName, nNumofFeature, nNumofInstance, nNumofValue);
    rawDataRead.ReadFromFileSparse(strTrainingFileName, nNumofFeature, v_v_DocVector, v_nLabel);
//    v_v_DocVector = vector<vector<svm_node> >(v_v_DocVector.begin(),v_v_DocVector.begin()+5000);
//    v_nLabel = vector<int>(v_nLabel.begin(), v_nLabel.begin()+5000);
    SvmProblem problem(v_v_DocVector, nNumofFeature, v_nLabel);
    start = clock();
    model.fit(problem, param);
    end = clock();
    printf("training time elapsed: %.2fs\n", (float) (end - start) / CLOCKS_PER_SEC);
    printf("total iteration time: %.2fs\n", nTimeOfLoop / 1e9);
    printf("read row time: %.2fs, read row count %ld\n", lGetHessianRowTime / 1e9, lGetHessianRowCounter);
    printf("cache hit time: %.2fs, cache hit count %ld\n", (lGetHessianRowTime - readRowTime) / 1e9, lGetHessianRowCounter - cacheMissCount);
    printf("cache miss time: %.2fs, cache miss count %ld\n", readRowTime / 1e9, cacheMissCount);
    printf("cache miss cuda memcpy time: %.2fs\n", cacheMissMemcpyTime / 1e9);
    printf("cache miss calculate hessian row time: %.2fs\n", (readRowTime - cacheMissMemcpyTime) / 1e9);
    printf("cache hit rate %.2f%%\n", (1 - (float) cacheMissCount / lGetHessianRowCounter) * 100);
    printf("ave time cache hit  %lf\nave time cache miss %lf\n",
           (lGetHessianRowTime-readRowTime)/1e9/(lGetHessianRowCounter-cacheMissCount), readRowTime/1e9/cacheMissCount);
}

svm_model trainBinarySVM(SvmProblem &problem, const SVMParam &param) {
    float_point pfCost = param.C;
    float_point pfGamma = param.gamma;
//    RBFKernelFunction f = RBFKernelFunction(param.gamma);
    DeviceHessianOnFly ops(problem, param.gamma);

    CLATCache cacheStrategy((const int &) problem.getNumOfSamples());
    cout << "using " << cacheStrategy.GetStrategy() << endl;
    CSMOSolver s(&ops, &cacheStrategy);
    s.problem = &problem;
    CSVMTrainer svmTrainer(&s);

    printf("cost %.2f\n", pfCost);
    gfNCost = pfCost;
    gfPCost = pfCost;

    //copy training information from input parameters
    const int *pnLabelAll = problem.v_nLabels.data();
    int nTotalNumofSamples = (int) problem.getNumOfSamples();

    /* allocate GPU device memory */
    //set default value at
    float_point *pfAlphaAll = new float_point[nTotalNumofSamples];
    float_point *pfYiGValueAll = new float_point[nTotalNumofSamples];
    for (int i = 0; i < nTotalNumofSamples; i++) {
        //initially, the values of alphas are 0s
        pfAlphaAll[i] = 0;
        //GValue is -y_i, as all alphas are 0s. YiGValue is always -1
        pfYiGValueAll[i] = -pnLabelAll[i];
    }

    //allocate GPU memory for part of samples that are used to perform training.
    float_point *pfDevAlphaSubset;
    float_point *pfDevYiGValueSubset;
    int *pnDevLabelSubset;

    //get size of training samples
    int nNumofTrainingSamples = nTotalNumofSamples;

    checkCudaErrors(hipMalloc((void **) &pfDevAlphaSubset, sizeof(float_point) * nNumofTrainingSamples));
    checkCudaErrors(hipMalloc((void **) &pfDevYiGValueSubset, sizeof(float_point) * nNumofTrainingSamples));
    checkCudaErrors(hipMalloc((void **) &pnDevLabelSubset, sizeof(int) * nNumofTrainingSamples));

    //copy training information to GPU for current training
    checkCudaErrors(hipMemcpy(pfDevAlphaSubset, pfAlphaAll,
                               sizeof(float_point) * nTotalNumofSamples, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(pfDevYiGValueSubset, pfYiGValueAll,
                               sizeof(float_point) * nTotalNumofSamples, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(pnDevLabelSubset, pnLabelAll,
                               sizeof(int) * nTotalNumofSamples, hipMemcpyHostToDevice));

    /************** train SVM model **************/
    svm_model model;
    model.param.C = param.C;
    model.param.gamma = param.gamma;
    svmTrainer.SetInvolveTrainingData(0, nNumofTrainingSamples - 1, -1, -1);
    bool bTrain = svmTrainer.TrainModel(model, pfDevYiGValueSubset, pfDevAlphaSubset,
                                        pnDevLabelSubset, nNumofTrainingSamples, NULL);
    if (bTrain == false) {
        cerr << "can't find an optimal classifier" << endl;
    }


    //free device memory
    checkCudaErrors(hipFree(pfDevAlphaSubset));
    checkCudaErrors(hipFree(pnDevLabelSubset));
    checkCudaErrors(hipFree(pfDevYiGValueSubset));

    delete[] pfAlphaAll;
    delete[] pfYiGValueAll;

    return model;
}

void evaluateSVMClassifier(SvmModel &model, string strTrainingFileName, int nNumofFeature) {
    vector<vector<svm_node> > v_v_DocVector;
    vector<int> v_nLabel;

    CDataIOOps rawDataRead;
    int nNumofInstance = 0;     //not used
    long long nNumofValue = 0;  //not used
    BaseLibSVMReader::GetDataInfo(strTrainingFileName, nNumofFeature, nNumofInstance, nNumofValue);
    rawDataRead.ReadFromFileSparse(strTrainingFileName, nNumofFeature, v_v_DocVector, v_nLabel);

    //perform svm classification

    int batchSize = 1000;
    int begin = 0;
    vector<int> predictLabels;
    clock_t start, end;
    start = clock();
    while (begin < v_v_DocVector.size()) {
        vector<vector<svm_node> > samples(v_v_DocVector.begin() + begin,
                                          v_v_DocVector.begin() + min(begin + batchSize, (int) v_v_DocVector.size()));
        vector<int> predictLabelPart = model.predict(samples, model.isProbability());
        predictLabels.insert(predictLabels.end(), predictLabelPart.begin(), predictLabelPart.end());
        begin += batchSize;
    }
    end = clock();
    int numOfCorrect = 0;
    for (int i = 0; i < v_v_DocVector.size(); ++i) {
        if (predictLabels[i] == v_nLabel[i])
            numOfCorrect++;
    }
    printf("training accuracy = %.2f%%(%d/%d)\n", numOfCorrect / (float) v_v_DocVector.size() * 100,
           numOfCorrect, (int) v_v_DocVector.size());
    printf("prediction time elapsed: %.2fs\n", (float) (end - start) / CLOCKS_PER_SEC);
}
