#include "hip/hip_runtime.h"
/*
 * testTrainer.cpp
 *
 *  Created on: 31/10/2013
 *      Author: Zeyi Wen
 */

#include "trainingFunction.h"
#include <sys/time.h>
#include "../svm-shared/Cache/cache.h"
#include "DataIOOps/DataIO.h"
#include "../svm-shared/HessianIO/deviceHessianOnFly.h"
#include "../SharedUtility/Timer.h"
#include "../SharedUtility/KeyValue.h"

void trainSVM(SVMParam &param, string strTrainingFileName, int nNumofFeature, SvmModel &model, bool evaluteTrainingError) {
    vector<vector<KeyValue> > v_v_Instance;
    vector<int> v_nLabel;

    int nNumofInstance = 0;     //not used
    long long nNumofValue = 0;  //not used
    BaseLibSVMReader::GetDataInfo(strTrainingFileName, nNumofFeature, nNumofInstance, nNumofValue);
	LibSVMDataReader drHelper;
	drHelper.ReadLibSVMAsSparse(v_v_Instance, v_nLabel, strTrainingFileName, nNumofFeature);
    SvmProblem problem(v_v_Instance, nNumofFeature, v_nLabel);
    ACCUMULATE_TIME(trainingTimer, model.fit(problem, param))
    PRINT_TIME("training", trainingTimer)
    PRINT_TIME("pre-computation kernel",preComputeTimer)
    PRINT_TIME("iteration",iterationTimer)
    PRINT_TIME("2 instances selection",selectTimer)
    PRINT_TIME("kernel calculation",calculateKernelTimer)
    PRINT_TIME("alpha updating",updateAlphaTimer)
    PRINT_TIME("g value updating time",updateGTimer)
    //evaluate training error
    if (evaluteTrainingError == true) {
        printf("Computing training accuracy...\n");
        evaluate(model, v_v_Instance, v_nLabel);
    }
}

void evaluateSVMClassifier(SvmModel &model, string strTrainingFileName, int nNumofFeature) {
    vector<vector<KeyValue> > v_v_Instance;
    vector<int> v_nLabel;

    int nNumofInstance = 0;     //not used
    long long nNumofValue = 0;  //not used
    BaseLibSVMReader::GetDataInfo(strTrainingFileName, nNumofFeature, nNumofInstance, nNumofValue);
	LibSVMDataReader drHelper;
	drHelper.ReadLibSVMAsSparse(v_v_Instance, v_nLabel, strTrainingFileName, nNumofFeature);

    //evaluate testing error
    evaluate(model, v_v_Instance, v_nLabel);
}

/**
 * @brief: evaluate the svm model, given some labeled instances.
 */
void evaluate(SvmModel &model, vector<vector<KeyValue> > &v_v_Instance, vector<int> &v_nLabel)
{
    //perform svm classification

    int batchSize = 2000;
    int begin = 0;
    vector<int> predictLabels;
    clock_t start, end;
    start = clock();
    while (begin < v_v_Instance.size()) {
        vector<vector<KeyValue> > samples(v_v_Instance.begin() + begin,
                                          v_v_Instance.begin() + min(begin + batchSize, (int) v_v_Instance.size()));
        vector<int> predictLabelPart = model.predict(samples, model.isProbability());
        predictLabels.insert(predictLabels.end(), predictLabelPart.begin(), predictLabelPart.end());
        begin += batchSize;
    }
    end = clock();
    int numOfCorrect = 0;
    for (int i = 0; i < v_v_Instance.size(); ++i) {
        if (predictLabels[i] == v_nLabel[i])
            numOfCorrect++;
    }
    printf("classifier accuracy = %.2f%%(%d/%d)\n", numOfCorrect / (float) v_v_Instance.size() * 100,
           numOfCorrect, (int) v_v_Instance.size());
    printf("prediction time elapsed: %.2fs\n", (float) (end - start) / CLOCKS_PER_SEC);
}
