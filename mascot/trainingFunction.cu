/*
 * testTrainer.cpp
 *
 *  Created on: 31/10/2013
 *      Author: Zeyi
 */

#include "trainingFunction.h"

#include<iostream>
#include<cassert>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <sys/time.h>
#include <sys/sysinfo.h>

#include "../svm-shared/gpu_global_utility.h"
#include "../svm-shared/constant.h"
#include "../svm-shared/HessianIO/baseHessian.h"
#include "../svm-shared/HessianIO/seqAccessor.h"
#include "../svm-shared/kernelCalculater/kernelCalculater.h"
#include "../svm-shared/svmTrainer.h"
#include "../svm-shared/smoSolver.h"
#include "../svm-shared/Cache/cache.h"
#include "DataIOOps/DataIO.h"
#include "DataIOOps/BaseLibsvmReader.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>

#include "svmProblem.h"
#include "../svm-shared/HessianIO/deviceHessianOnFly.h"

using std::cout;
using std::endl;

void trainSVM(SVMParam &param, string strTrainingFileName, int nNumofFeature, SvmModel &model) {

    vector<vector<svm_node> > v_v_DocVector;
    vector<int> v_nLabel;

    CDataIOOps rawDataRead;
    int nNumofInstance = 0;     //not used
    long long nNumofValue = 0;  //not used
    BaseLibSVMReader::GetDataInfo(strTrainingFileName, nNumofFeature, nNumofInstance, nNumofValue);
    rawDataRead.ReadFromFileSparse(strTrainingFileName, nNumofFeature, v_v_DocVector, v_nLabel);
    SvmProblem problem(v_v_DocVector, v_nLabel);
    model.fit(problem, param);
}

svm_model trainBinarySVM(SvmProblem &problem, const SVMParam &param) {
    float_point pfCost = param.C;
    float_point pfGamma = param.gamma;
//    RBFKernelFunction f = RBFKernelFunction(param.gamma);
    DeviceHessianOnFly ops(problem,param.gamma);

    CLATCache cacheStrategy((const int &) problem.getNumOfSamples());
    cout << "using " << cacheStrategy.GetStrategy() << endl;
    CSMOSolver s(&ops, &cacheStrategy);
    CSVMTrainer svmTrainer(&s);

    gfNCost = pfCost;
    gfPCost = pfCost;

    //copy training information from input parameters
    const int *pnLabelAll = problem.v_nLabels.data();
    int nTotalNumofSamples = (int) problem.getNumOfSamples();

    /* allocate GPU device memory */
    //set default value at
    float_point *pfAlphaAll = new float_point[nTotalNumofSamples];
    float_point *pfYiGValueAll = new float_point[nTotalNumofSamples];
    for (int i = 0; i < nTotalNumofSamples; i++) {
        //initially, the values of alphas are 0s
        pfAlphaAll[i] = 0;
        //GValue is -y_i, as all alphas are 0s. YiGValue is always -1
        pfYiGValueAll[i] = -pnLabelAll[i];
    }

    //allocate GPU memory for part of samples that are used to perform training.
    float_point *pfDevAlphaSubset;
    float_point *pfDevYiGValueSubset;
    int *pnDevLabelSubset;

    //get size of training samples
    int nNumofTrainingSamples = nTotalNumofSamples;

    checkCudaErrors(hipMalloc((void **) &pfDevAlphaSubset, sizeof(float_point) * nNumofTrainingSamples));
    checkCudaErrors(hipMalloc((void **) &pfDevYiGValueSubset, sizeof(float_point) * nNumofTrainingSamples));
    checkCudaErrors(hipMalloc((void **) &pnDevLabelSubset, sizeof(int) * nNumofTrainingSamples));

    //copy training information to GPU for current training
    checkCudaErrors(hipMemcpy(pfDevAlphaSubset, pfAlphaAll,
                               sizeof(float_point) * nTotalNumofSamples, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(pfDevYiGValueSubset, pfYiGValueAll,
                               sizeof(float_point) * nTotalNumofSamples, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(pnDevLabelSubset, pnLabelAll,
                               sizeof(int) * nTotalNumofSamples, hipMemcpyHostToDevice));

    /************** train SVM model **************/
    svm_model model;
    model.param.C = param.C;
    model.param.gamma = param.gamma;
    svmTrainer.SetInvolveTrainingData(0, nNumofTrainingSamples - 1, -1, -1);
    bool bTrain = svmTrainer.TrainModel(model, pfDevYiGValueSubset, pfDevAlphaSubset,
                                        pnDevLabelSubset, nNumofTrainingSamples, NULL);
    if (bTrain == false) {
        cerr << "can't find an optimal classifier" << endl;
    }


    //free device memory
    checkCudaErrors(hipFree(pfDevAlphaSubset));
    checkCudaErrors(hipFree(pnDevLabelSubset));
    checkCudaErrors(hipFree(pfDevYiGValueSubset));

    delete[] pfAlphaAll;
    delete[] pfYiGValueAll;

    return model;
}

void evaluateSVMClassifier(SvmModel &model, string strTrainingFileName, int nNumofFeature)
{
    vector<vector<svm_node> > v_v_DocVector;
    vector<int> v_nLabel;

    CDataIOOps rawDataRead;
    int nNumofInstance = 0;     //not used
    long long nNumofValue = 0;  //not used
    BaseLibSVMReader::GetDataInfo(strTrainingFileName, nNumofFeature, nNumofInstance, nNumofValue);
    rawDataRead.ReadFromFileSparse(strTrainingFileName, nNumofFeature, v_v_DocVector, v_nLabel);

    //perform svm classification

    vector<int> predictLabels = model.predict(v_v_DocVector, model.isProbability());
    int numOfCorrect = 0;
    for (int i = 0; i < v_v_DocVector.size(); ++i) 
    {
        if (predictLabels[i] == v_nLabel[i])
            numOfCorrect++;
    }
    printf("training accuracy = %.2f%%(%d/%d)\n", numOfCorrect / (float) v_v_DocVector.size()*100, 
            numOfCorrect, (int) v_v_DocVector.size());
}
