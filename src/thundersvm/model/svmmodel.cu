#include "hip/hip_runtime.h"
//
// Created by jiashuai on 17-9-21.
//

#include <thundersvm/kernel/smo_kernel.h>
#include <thrust/sort.h>
#include <thrust/system/cuda/detail/par.h>
#include <thundersvm/model/svmmodel.h>

SvmModel::SvmModel(DataSet &dataSet, const SvmParam &svmParam) : dataSet(dataSet), svmParam(svmParam) {

}

int SvmModel::max2power(int n) const {
    return int(pow(2, floor(log2f(float(n)))));
}

void
SvmModel::smo_solver(const KernelMatrix &k_mat, const SyncData<int> &y, SyncData<real> &alpha, real &rho,
                     SyncData<real> &init_f, real eps, real C, int ws_size) {
    uint n_instances = k_mat.m();
    uint q = ws_size / 2;

    SyncData<int> working_set(ws_size);
    SyncData<int> working_set_first_half(q);
    SyncData<int> working_set_last_half(q);
    working_set_first_half.set_device_data(working_set.device_data());
    working_set_last_half.set_device_data(&working_set.device_data()[q]);
    working_set_first_half.set_host_data(working_set.host_data());
    working_set_last_half.set_host_data(&working_set.host_data()[q]);

    SyncData<real> f(n_instances);
    SyncData<int> f_idx(n_instances);
    SyncData<int> f_idx2sort(n_instances);
    SyncData<real> f_val2sort(n_instances);
    SyncData<real> alpha_diff(ws_size);
    SyncData<real> diff_and_bias(2);

    SyncData<real> k_mat_rows(ws_size * k_mat.m());
    SyncData<real> k_mat_rows_first_half(q * k_mat.m());
    SyncData<real> k_mat_rows_last_half(q * k_mat.m());
    k_mat_rows_first_half.set_device_data(k_mat_rows.device_data());
    k_mat_rows_last_half.set_device_data(&k_mat_rows.device_data()[q * k_mat.m()]);
    CHECK_EQ(init_f.count(), n_instances);
    f.copy_from(init_f);
    for (int i = 0; i < n_instances; ++i) {
        f_idx[i] = i;
    }
    alpha.mem_set(0);
    LOG(INFO) << "training start";
    for (int iter = 1;; ++iter) {
        //select working set
        f_idx2sort.copy_from(f_idx);
        f_val2sort.copy_from(f);
        thrust::sort_by_key(thrust::cuda::par, f_val2sort.device_data(), f_val2sort.device_data() + n_instances,
                            f_idx2sort.device_data(), thrust::less<real>());
        vector<int> ws_indicator(n_instances, 0);
        if (1 == iter) {
            select_working_set(ws_indicator, f_idx2sort, y, alpha, working_set);
            k_mat.get_rows(working_set, k_mat_rows);
        } else {
            working_set_first_half.copy_from(working_set_last_half);
            for (int i = 0; i < q; ++i) {
                ws_indicator[working_set[i]] = 1;
            }
            select_working_set(ws_indicator, f_idx2sort, y, alpha, working_set_last_half);
            k_mat_rows_first_half.copy_from(k_mat_rows_last_half);
            k_mat.get_rows(working_set_last_half, k_mat_rows_last_half);
        }

        //local smo
        size_t smem_size = ws_size * sizeof(real) * 3 + 2 * sizeof(float);
        localSMO << < 1, ws_size, smem_size >> >
                                  (y.device_data(), f.device_data(), alpha.device_data(), alpha_diff.device_data(),
                                          working_set.device_data(), ws_size, C, k_mat_rows.device_data(), n_instances,
                                          eps, diff_and_bias.device_data());
        LOG_EVERY_N(10, INFO) << "diff=" << diff_and_bias[0];
        if (diff_and_bias[0] < eps) {
            rho = diff_and_bias[1];
            break;
        }

        //update f
        SAFE_KERNEL_LAUNCH(update_f, f.device_data(), ws_size, alpha_diff.device_data(), k_mat_rows.device_data(),
                           n_instances);
    }
}

void
SvmModel::select_working_set(vector<int> &ws_indicator, const SyncData<int> &f_idx2sort, const SyncData<int> &y,
                             const SyncData<real> &alpha, SyncData<int> &working_set) {
    int n_instances = ws_indicator.size();
    int p_left = 0;
    int p_right = n_instances - 1;
    int n_selected = 0;
    const int *index = f_idx2sort.host_data();
    while (n_selected < working_set.count()) {
        int i;
        if (p_left < n_instances) {
            i = index[p_left];
            while (ws_indicator[i] == 1 || !(y[i] > 0 && alpha[i] < svmParam.C || y[i] < 0 && alpha[i] > 0)) {
                p_left++;
                if (p_left == n_instances) break;
                i = index[p_left];
            }
            if (p_left < n_instances) {
                working_set[n_selected++] = i;
                ws_indicator[i] = 1;
            }
        }
        if (p_right >= 0) {
            i = index[p_right];
            while ((ws_indicator[i] == 1 || !(y[i] > 0 && alpha[i] > 0 || y[i] < 0 && alpha[i] < svmParam.C))) {
                p_right--;
                if (p_right == -1) break;
                i = index[p_right];
            }
            if (p_right >= 0) {
                working_set[n_selected++] = i;
                ws_indicator[i] = 1;
            }
        }

    }
}
