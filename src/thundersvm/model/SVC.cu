#include "hip/hip_runtime.h"
//
// Created by jiashuai on 17-9-21.
//
#include <thundersvm/kernel/smo_kernel.h>
#include "thundersvm/model/SVC.h"
#include "thrust/sort.h"
#include "thrust/system/cuda/execution_policy.h"

SVC::SVC(DataSet &dataSet, const SvmParam &svmParam) : SvmModel(dataSet, svmParam) {
    n_classes = dataSet.n_classes();
    n_binary_models = n_classes * (n_classes - 1) / 2;
    rho.resize(n_binary_models);
    sv_index.resize(n_binary_models);
    coef.resize(n_binary_models);
}

void SVC::train() {
    int k = 0;
    for (int i = 0; i < n_classes; ++i) {
        for (int j = i + 1; j < n_classes; ++j) {
            DataSet::node2d ins = dataSet.instances(i, j);
            size_t subproblem_size = ins.size();
            SyncData<int> y(subproblem_size);
            SyncData<real> alpha(subproblem_size);
            real rho;
            alpha.mem_set(0);
            for (int l = 0; l < dataSet.count()[i]; ++l) {
                y[l] = +1;
            }
            for (int l = 0; l < dataSet.count()[j]; ++l) {
                y[dataSet.count()[i] + l] = -1;
            }
            KernelMatrix k_mat(ins, dataSet.n_features(), svmParam.gamma);
            smo_solver(k_mat, y, alpha, rho, 0.001, svmParam.C);
            record_binary_model(k, alpha, y, rho, dataSet.original_index(i, j));
            k++;
        }
    }
}

void SVC::predict(DataSet &dataSet) {

}

void SVC::save_to_file(string path) {

}

void SVC::load_from_file(string path) {

}

void SVC::smo_solver(const KernelMatrix &k_mat, const SyncData<int> &y, SyncData<real> &alpha, real &rho, real eps,
                     real C) {
//    TIMED_FUNC(timer_obj);
    uint n_instances = k_mat.m();
    SyncData<real> f(n_instances);
//    LOG(INFO)<<min(10.,ceil(log2(float(n_instances))));
    uint ws_size = 1024;
    uint q = ws_size / 2;
    SyncData<int> working_set(ws_size);
    SyncData<int> f_idx(n_instances);
    SyncData<int> f_idx2sort(n_instances);
    SyncData<real> f_val2sort(n_instances);
    SyncData<real> alpha_diff(ws_size);
    SyncData<real> k_mat_rows(ws_size * k_mat.m());
    SyncData<real> diff_and_bias(2);
    for (int i = 0; i < n_instances; ++i) {
        f.host_data()[i] = -y.host_data()[i];
        f_idx.host_data()[i] = i;
    }
    alpha.mem_set(0);
    LOG(INFO) << "training start";
    for (int iter = 1;; ++iter) {
        //select working set
        f_idx2sort.copy_from(f_idx);
        f_val2sort.copy_from(f);
        thrust::sort_by_key(thrust::cuda::par, f_val2sort.device_data(), f_val2sort.device_data() + n_instances,
                            f_idx2sort.device_data(), thrust::less<real>());
        int *ws;
        vector<int> ws_indicator(n_instances, 0);
        if (1 == iter) {
            ws = working_set.host_data();
            q = ws_size;
        } else {
            q = ws_size / 2;
            working_set.copy_from(working_set.device_data() + q, q);
            ws = working_set.host_data() + q;
            for (int i = 0; i < q; ++i) {
                ws_indicator[working_set[i]] = 1;
            }
        }
        int p_left = 0;
        int p_right = n_instances - 1;
        int n_selected = 0;
        const int *index = f_idx2sort.host_data();
        while (n_selected < q) {
            int i;
            if (p_left < n_instances) {
                i = index[p_left];
                while (ws_indicator[i] == 1 || !(y[i] > 0 && alpha[i] < C || y[i] < 0 && alpha[i] > 0)) {
                    p_left++;
                    if (p_left == n_instances) break;
                    i = index[p_left];
                }
                if (p_left < n_instances) {
                    ws[n_selected++] = i;
                    ws_indicator[i] = 1;
                }
            }
            if (p_right >= 0) {
                i = index[p_right];
                while ((ws_indicator[i] == 1 || !(y[i] > 0 && alpha[i] > 0 || y[i] < 0 && alpha[i] < C))) {
                    p_right--;
                    if (p_right == -1) break;
                    i = index[p_right];
                }
                if (p_right >= 0) {
                    ws[n_selected++] = i;
                    ws_indicator[i] = 1;
                }
            }
        }

        //precompute kernel
        working_set.to_device();
        k_mat.get_rows(&working_set, &k_mat_rows);
        //local smo
        size_t smem_size = ws_size * sizeof(real) * 3 + 2 * sizeof(float);
        localSMO << < 1, ws_size, smem_size >> >
                                  (y.device_data(), f.device_data(), alpha.device_data(), alpha_diff.device_data(),
                                          working_set.device_data(), ws_size, C, k_mat_rows.device_data(), n_instances,
                                          eps, diff_and_bias.device_data());
        LOG_EVERY_N(10, INFO) << "diff=" << diff_and_bias[0];
        if (diff_and_bias[0] < eps) {
            rho = diff_and_bias[1];
            break;
        }
        //update f
        update_f << < NUM_BLOCKS, BLOCK_SIZE >> >
                                  (f.device_data(), ws_size, alpha_diff.device_data(), k_mat_rows.device_data(), n_instances);
    }
}

void SVC::record_binary_model(int k, const SyncData<real> &alpha, const SyncData<int> &y, real rho,
                              const vector<int> &original_index) {
    int n_sv = 0;
    for (int i = 0; i < alpha.count(); ++i) {
        if (alpha[i] != 0) {
            coef[k].push_back(alpha[i] * y[i]);
            if (sv_index_map.find(original_index[i]) == sv_index_map.end()) {
                sv_index_map[original_index[i]] = sv_index_map.size();
                sv.push_back(dataSet.instances()[original_index[i]]);
            }
            sv_index[k].push_back(sv_index_map[original_index[i]]);
            n_sv++;
        }
    }
    this->rho[k] = rho;
    LOG(INFO) << "rho=" << rho;
    LOG(INFO) << "#SV=" << n_sv;
}

