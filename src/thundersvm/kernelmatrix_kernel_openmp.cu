#include "hip/hip_runtime.h"
#include "thundersvm/kernelmatrix_kernel_openmp.h"
#include <iostream>
void kernel_get_working_set_ins_openmp(const real *val, const int *col_ind, const int *row_ptr, const int *data_row_idx,
                           real *data_rows,
                           int m) {
#pragma omp parallel for
    for(int i = 0; i < m; i++) {
        int row = data_row_idx[i];
	//#pragma omp parallel for
        for (int j = row_ptr[row]; j < row_ptr[row + 1]; ++j) {
            int col = col_ind[j];
            data_rows[col * m + i] = val[j]; // row-major for cuSPARSE
        }
    }
}

void kernel_RBF_kernel_openmp(const real *self_dot0, const real *self_dot1, real *dot_product, int m, int n, real gamma) {
    //m rows of kernel matrix, where m is the working set size; n is the number of training instances
#pragma omp parallel for
    for(int idx = 0; idx < m * n; idx++) {
        int i = idx / n;//i is row id
        int j = idx % n;//j is column id
        dot_product[idx] = expf(-(self_dot0[i] + self_dot1[j] - dot_product[idx] * 2) * gamma);
    }
}

void kernel_RBF_kernel_openmp(const int *self_dot0_idx, const real *self_dot1, real *dot_product, int m, int n, real gamma) {
    //compute m rows of kernel matrix, where m is the working set size and n is the number of training instances, according to idx
#pragma omp parallel for
    for(int idx = 0; idx < m * n; idx++){
        int i = idx / n;//i is row id
        int j = idx % n;//j is column id
        dot_product[idx] = expf(-(self_dot1[self_dot0_idx[i]] + self_dot1[j] - dot_product[idx] * 2) * gamma);
    }
}


void kernel_sum_kernel_values_openmp(const real *k_mat, int n_instances, int n_sv_unique, int n_bin_models,
                                         const int *sv_index, const real *coef, const int *sv_start,
                                         const int *sv_count,
                                         const real *rho, real *dec_values) {//compute decision values for n_instances
 
#pragma omp parallel for
    for(int idx = 0; idx < n_instances * n_bin_models; idx++){
        //one iteration uses a binary svm model to predict a decision value of an instance.
        //#ifndef _OPENMP
            //std::cout<<"no openmp"<<std::endl;
        //#endif
        int ins_id = idx / n_bin_models;
        int model_id = idx % n_bin_models;
        real sum = 0;
        const real *kernel_row = k_mat + ins_id * n_sv_unique;//kernel values of this instance
        int si = sv_start[model_id];
        int ci = sv_count[model_id];
        #pragma omp parallel for reduction(+:sum)
        for (int i = 0; i < ci; ++i) {//TODO: improve by parallelism
            sum += coef[si + i] * kernel_row[sv_index[si + i]];//sv_index maps uncompressed sv idx to compressed sv idx.
        }
        dec_values[idx] = sum - rho[model_id];
    }

}

void kernel_poly_kernel_openmp(real *dot_product, real gamma, real coef0, int degree, int mn) {
#pragma omp parallel for
    for(int idx = 0; idx < mn; idx++){
        dot_product[idx] = powf(gamma * dot_product[idx] + coef0, degree);
    }
}

void kernel_sigmoid_kernel_openmp(real *dot_product, real gamma, real coef0, int mn) {
    //KERNEL_LOOP(idx, mn) {
#pragma omp parallel for
    for(int idx = 0; idx < mn; idx++){
        dot_product[idx] = tanhf(gamma * dot_product[idx] + coef0);
    }
}
