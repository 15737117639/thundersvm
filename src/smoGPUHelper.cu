#include "hip/hip_runtime.h"

#include "smoGPUHelper.h"
//#include "devUtility2.h"
#include <float.h>

//-----------------------------------

__device__ void GetMinValueOriginal(float_point *pfValues, int *pnKey, int nNumofBlock)
{
	/*if(1024 < BLOCK_SIZE)
	{
		printf("block size is two large!\n");
		return;
	}*/
	//Reduce by a factor of 2, and minimize step size
	int nTid = threadIdx.x;
	int compOffset;

	if(BLOCK_SIZE == 128)
	{
		compOffset = nTid + 64;
		if(nTid < 64)
		{
			if(compOffset < nNumofBlock)
			{
				if(pfValues[compOffset] < pfValues[nTid])
				{
					pnKey[nTid] = pnKey[compOffset];
					pfValues[nTid] = pfValues[compOffset];
				}
			}
		}
		//synchronise threads to avoid read dirty value (dirty read may happen if two steps reduction, say 32 and 16, run simultaneously)
		__syncthreads();
	}
		compOffset = nTid + 32;
		if(nTid < 32 && (compOffset < nNumofBlock))
		{
			if(pfValues[compOffset] < pfValues[nTid])
			{
				pnKey[nTid] = pnKey[compOffset];
				pfValues[nTid] = pfValues[compOffset];
			}
		}
		//synchronise threads to avoid read dirty value (dirty read may happen if two steps reduction, say 32 and 16, run simultaneously)
		__syncthreads();

		compOffset = nTid + 16;
		if(nTid < 16 && (compOffset < nNumofBlock))
		{
			if(pfValues[compOffset] < pfValues[nTid])
			{
				pnKey[nTid] = pnKey[compOffset];
				pfValues[nTid] = pfValues[compOffset];
			}
		}

		compOffset = nTid + 8;
		if(nTid < 8 && (compOffset < nNumofBlock))
		{
			if(pfValues[compOffset] < pfValues[nTid])
			{
				pnKey[nTid] = pnKey[compOffset];
				pfValues[nTid] = pfValues[compOffset];
			}
		}

		compOffset = nTid + 4;
		if(nTid < 4 && (compOffset < nNumofBlock))
		{
			if(pfValues[compOffset] < pfValues[nTid])
			{
				pnKey[nTid] = pnKey[compOffset];
				pfValues[nTid] = pfValues[compOffset];
			}
		}

		compOffset = nTid + 2;
		if(nTid < 2 && (compOffset < nNumofBlock))
		{
			if(pfValues[compOffset] < pfValues[nTid])
			{
				pnKey[nTid] = pnKey[compOffset];
				pfValues[nTid] = pfValues[compOffset];
			}
		}

		compOffset = nTid + 1;
		if(nTid < 1 && (compOffset < nNumofBlock))
		{
			if(pfValues[compOffset] < pfValues[nTid])
			{
				pnKey[nTid] = pnKey[compOffset];
				pfValues[nTid] = pfValues[compOffset];
			}
		}

}

__device__ void GetMinValueOriginal(float_point *pfValues, int nNumofBlock)
{
	/*if(1024 < BLOCK_SIZE)
	{
		printf("block size is two large!\n");
		return;
	}*/
	//Reduce by a factor of 2, and minimize step size
	int nTid = threadIdx.x;
	int compOffset;

	if(BLOCK_SIZE == 128)
	{
		compOffset = nTid + 64;
		if(nTid < 64 && (compOffset < nNumofBlock))
		{
			if(pfValues[compOffset] < pfValues[nTid])
			{
				pfValues[nTid] = pfValues[compOffset];
			}
		}
//		else
//			return;
		//synchronise threads to avoid read dirty value (dirty read may happen if two steps reduction, say 32 and 16, run simultaneously)
		__syncthreads();
	}
		compOffset = nTid + 32;
		if(nTid < 32 && (compOffset < nNumofBlock))
		{
			if(pfValues[compOffset] < pfValues[nTid])
			{
				pfValues[nTid] = pfValues[compOffset];
			}
		}
//		else
//			return;
		//synchronise threads to avoid read dirty value (dirty read may happen if two steps reduction, say 32 and 16, run simultaneously)
		__syncthreads();

		compOffset = nTid + 16;
		if(nTid < 16 && (compOffset < nNumofBlock))
		{
			if(pfValues[compOffset] < pfValues[nTid])
			{
				pfValues[nTid] = pfValues[compOffset];
			}
		}
//		else
//			return;

		compOffset = nTid + 8;
		if(nTid < 8 && (compOffset < nNumofBlock))
		{
			if(pfValues[compOffset] < pfValues[nTid])
			{
				pfValues[nTid] = pfValues[compOffset];
			}
		}
//		else
//			return;

		compOffset = nTid + 4;
		if(nTid < 4 && (compOffset < nNumofBlock))
		{
			if(pfValues[compOffset] < pfValues[nTid])
			{
				pfValues[nTid] = pfValues[compOffset];
			}
		}
//		else
//			return;


		compOffset = nTid + 2;
		if(nTid < 2 && (compOffset < nNumofBlock))
		{
			if(pfValues[compOffset] < pfValues[nTid])
			{
				pfValues[nTid] = pfValues[compOffset];
			}
		}
//		else
//			return;

		compOffset = nTid + 1;
		if(nTid < 1 && (compOffset < nNumofBlock))
		{
			if(pfValues[compOffset] < pfValues[nTid])
			{
				pfValues[nTid] = pfValues[compOffset];
			}
		}

}


/* *
 /*
 * @brief: use reducer to get the minimun value in parallel
 * @param: pfValues: a pointer to a set of data
 * @param: pnKey:	 a pointer to the index of the set of data. It's for getting the location of min.
 */
__device__ void GetMinValue(float_point *pfValues, int *pnKey, int nNumofBlock)
{
	/*if(1024 < BLOCK_SIZE)
	{
		printf("block size is two large!\n");
		return;
	}*/
	//Reduce by a factor of 2, and minimize step size
	int nTid = threadIdx.x;
	int compOffset;
	float_point fValue1, fValue2;
	fValue1 = pfValues[nTid];

	if(BLOCK_SIZE == 128)
	{
		compOffset = nTid + 64;
		if(nTid < 64)
		{
			if(compOffset < nNumofBlock)
			{
				fValue2 = pfValues[compOffset];
				if(fValue2 < fValue1)
				{
					pnKey[nTid] = pnKey[compOffset];
					pfValues[nTid] = fValue2;
					fValue1 = fValue2;
				}
			}
		}
		//synchronise threads to avoid read dirty value (dirty read may happen if two steps reduction, say 32 and 16, run simultaneously)
		__syncthreads();
	}
		compOffset = nTid + 32;
		if(nTid < 32 && (compOffset < nNumofBlock))
		{
			fValue2 = pfValues[compOffset];
			if(fValue2 < fValue1)
			{
				pnKey[nTid] = pnKey[compOffset];
				pfValues[nTid] = fValue2;
				fValue1 = fValue2;
			}
		}
		//synchronise threads to avoid read dirty value (dirty read may happen if two steps reduction, say 32 and 16, run simultaneously)
		__syncthreads();

		compOffset = nTid + 16;
		if(nTid < 16 && (compOffset < nNumofBlock))
		{
			fValue2 = pfValues[compOffset];
			if(fValue2 < fValue1)
			{
				pnKey[nTid] = pnKey[compOffset];
				pfValues[nTid] = fValue2;
				fValue1 = fValue2;
			}
		}

		compOffset = nTid + 8;
		if(nTid < 8 && (compOffset < nNumofBlock))
		{
			fValue2 = pfValues[compOffset];
			if(fValue2 < fValue1)
			{
				pnKey[nTid] = pnKey[compOffset];
				pfValues[nTid] = fValue2;
				fValue1 = fValue2;
			}
		}

		compOffset = nTid + 4;
		if(nTid < 4 && (compOffset < nNumofBlock))
		{
			fValue2 = pfValues[compOffset];
			if(fValue2 < fValue1)
			{
				pnKey[nTid] = pnKey[compOffset];
				pfValues[nTid] = fValue2;
				fValue1 = fValue2;
			}
		}

		compOffset = nTid + 2;
		if(nTid < 2 && (compOffset < nNumofBlock))
		{
			fValue2 = pfValues[compOffset];
			if(fValue2 < fValue1)
			{
				pnKey[nTid] = pnKey[compOffset];
				pfValues[nTid] = fValue2;
				fValue1 = fValue2;
			}
		}

		compOffset = nTid + 1;
		if(nTid < 1 && (compOffset < nNumofBlock))
		{
			fValue2 = pfValues[compOffset];
			if(fValue2 < fValue1)
			{
				pnKey[nTid] = pnKey[compOffset];
				pfValues[nTid] = fValue2;
				//fValue1 = fValue2;
			}
		}

}

__device__ void GetMinValue(float_point *pfValues, int nNumofBlock)
{
	/*if(1024 < BLOCK_SIZE)
	{
		printf("block size is two large!\n");
		return;
	}*/
	//Reduce by a factor of 2, and minimize step size
	int nTid = threadIdx.x;
	int compOffset;
	float_point fValue1, fValue2;
	fValue1 = pfValues[nTid];

	if(BLOCK_SIZE == 128)
	{
		compOffset = nTid + 64;
		if(nTid < 64 && (compOffset < nNumofBlock))
		{
			fValue2 = pfValues[compOffset];
			fValue1 = (fValue2 < fValue1) ? fValue2 : fValue1;
			pfValues[nTid] = fValue1;
		}
//		else if(nTid >= 64)
//			return;
		//synchronise threads to avoid read dirty value (dirty read may happen if two steps reduction, say 32 and 16, run simultaneously)
		__syncthreads();
	}
		compOffset = nTid + 32;
		if(nTid < 32 && (compOffset < nNumofBlock))
		{
			fValue2 = pfValues[compOffset];
			fValue1 = (fValue2 < fValue1) ? fValue2 : fValue1;
			pfValues[nTid] = fValue1;
		}
//		else if(nTid >= 32)
//			return;
		//synchronise threads to avoid read dirty value (dirty read may happen if two steps reduction, say 32 and 16, run simultaneously)
		__syncthreads();

		compOffset = nTid + 16;
		if(nTid < 16 && (compOffset < nNumofBlock))
		{
			fValue2 = pfValues[compOffset];
			fValue1 = (fValue2 < fValue1) ? fValue2 : fValue1;
			pfValues[nTid] = fValue1;
		}
//		else
//			return;

		compOffset = nTid + 8;
		if(nTid < 8 && (compOffset < nNumofBlock))
		{
			fValue2 = pfValues[compOffset];
			fValue1 = (fValue2 < fValue1) ? fValue2 : fValue1;
			pfValues[nTid] = fValue1;
		}
//		else
//			return;

		compOffset = nTid + 4;
		if(nTid < 4 && (compOffset < nNumofBlock))
		{
			fValue2 = pfValues[compOffset];
			fValue1 = (fValue2 < fValue1) ? fValue2 : fValue1;
			pfValues[nTid] = fValue1;
		}
//		else
//			return;


		compOffset = nTid + 2;
		if(nTid < 2 && (compOffset < nNumofBlock))
		{
			fValue2 = pfValues[compOffset];
			fValue1 = (fValue2 < fValue1) ? fValue2 : fValue1;
			pfValues[nTid] = fValue1;
		}
//		else
//			return;

		compOffset = nTid + 1;
		if(nTid < 1 && (compOffset < nNumofBlock))
		{
			fValue2 = pfValues[compOffset];
			pfValues[nTid] = (fValue2 < fValue1) ? fValue2 : fValue1;
		}

}


/* *
 /*
 * @brief: use reducer to get the minimun value in parallel
 * @param: pfValues: a pointer to a set of data
 * @param: pnKey:	 a pointer to the index of the set of data. It's for getting the location of min.
 */
__device__ void GetBigMinValue(float_point *pfValues, int *pnKey)
{
	/*if(1024 < BLOCK_SIZE)
	{
		printf("block size is two large!\n");
		return;
	}*/
	//Reduce by a factor of 2, and minimize step size
	int nTid = threadIdx.x;
	int compOffset;
	float_point fValue1, fValue2;
	fValue1 = pfValues[nTid];
	int nNumofBlock = BLOCK_SIZE * TASK_OF_THREAD;

	for(int i = BLOCK_SIZE * (TASK_OF_THREAD - 1); i >= BLOCK_SIZE; i -= BLOCK_SIZE)
	{
		compOffset = nTid + i;
		if(nTid < i && (compOffset < nNumofBlock))
		{
			fValue2 = pfValues[compOffset];
			if(fValue2 < fValue1)
			{
				pnKey[nTid] = pnKey[compOffset];
				pfValues[nTid] = fValue2;
				fValue1 = fValue2;
			}
		}
		//synchronise threads to avoid read dirty value (dirty read may happen if two steps reduction, say 32 and 16, run simultaneously)
		__syncthreads();
	}

	GetMinValue(pfValues, pnKey, BLOCK_SIZE);
	/*#pragma unroll UNROLL_REDUCE
	for(int i = (BLOCK_SIZE / 2); i >= 1; i /= 2)
	{
		compOffset = nTid + i;
		if(nTid < i)
		{
			if(compOffset < nNumofBlock)
			{
				fValue2 = pfValues[compOffset];
				if(fValue2 < fValue1)
				{
					pnKey[nTid] = pnKey[compOffset];
					pfValues[nTid] = fValue2;
					fValue1 = fValue2;
				}
			}
		}
		else
			return;
		//synchronise threads to avoid read dirty value (dirty read may happen if two steps reduction, say 32 and 16, run simultaneously)
		__syncthreads();
	}*/

}

__device__ void GetBigMinValue(float_point *pfValues)
{
	int nTid = threadIdx.x;
	int compOffset;
	float_point fValue1, fValue2;
	fValue1 = pfValues[nTid];
	int nNumofBlock = BLOCK_SIZE * TASK_OF_THREAD;

	for(int i = BLOCK_SIZE * (TASK_OF_THREAD - 1); i >= BLOCK_SIZE; i -= BLOCK_SIZE)
	{
		compOffset = nTid + i;
		if(nTid < i && (compOffset < nNumofBlock))
		{
			fValue2 = pfValues[compOffset];
			fValue1 = (fValue2 < fValue1) ? fValue2 : fValue1;
			pfValues[nTid] = fValue1;
		}
		//synchronise threads to avoid read dirty value (dirty read may happen if two steps reduction, say 32 and 16, run simultaneously)
		__syncthreads();
	}

	GetMinValue(pfValues, BLOCK_SIZE);
}

//-----------------------------------


/* *
 /*
 * @brief: kernel funciton for getting minimum value within a block
 * @param: pfYiFValue: a set of value = y_i * gradient of subjective function
 * @param: pfAlpha:	   a set of alpha related to training samples
 * @param: pnLabel:	   a set of labels related to training samples
 * @param: nNumofTrainingSamples: the number of training samples
 * @param: pfBlockMin: the min value of this block (function result)
 * @param: pnBlockMinGlobalKey: the index of the min value of this block
 */
__global__ void GetBlockMinYiGValue(float_point *pfYiFValue, float_point *pfAlpha, int *pnLabel, float_point fPCost,
									int nNumofTraingSamples, float_point *pfBlockMin, int *pnBlockMinGlobalKey)
{
	__shared__ float_point fTempLocalYiFValue[BLOCK_SIZE];
	__shared__ int nTempLocalKeys[BLOCK_SIZE];

	int nGlobalIndex;
	int nThreadId = threadIdx.x;
	nGlobalIndex = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;//global index for thread

	float_point fAlpha;
	int nLabel;
	fAlpha = pfAlpha[nGlobalIndex];
	nLabel = pnLabel[nGlobalIndex];
	fTempLocalYiFValue[nThreadId] = FLT_MAX;
	//fill yi*GValue in a block
	if(nGlobalIndex < nNumofTraingSamples && ((nLabel > 0 && fAlpha < fPCost) || (nLabel < 0 && fAlpha > 0)))
	{
		//I_0 is (fAlpha > 0 && fAlpha < fCostP). This condition is covered by the following condition
		//index set I_up
		fTempLocalYiFValue[nThreadId] = pfYiFValue[nGlobalIndex];
		nTempLocalKeys[nThreadId] = nGlobalIndex;
	}
	__syncthreads();	//synchronize threads within a block, and start to do reduce

	GetMinValueOriginal(fTempLocalYiFValue, nTempLocalKeys, blockDim.x);

	if(nThreadId == 0)
	{
		int nBlockId =  blockIdx.y * gridDim.x + blockIdx.x;
		pfBlockMin[nBlockId] = fTempLocalYiFValue[0];
		pnBlockMinGlobalKey[nBlockId] = nTempLocalKeys[0];
	}
}

/*
 * @brief: for selecting the second sample to optimize
 * @param: pfYiFValue: the gradient of data samples
 * @param: pfAlpha: alpha values for samples
 * @param: fNCost: the cost of negative sample (i.e., the C in SVM)
 * @param: pfDiagHessian: the diagonal of Hessian Matrix
 * @param: pfHessianRow: a Hessian row of sample one
 * @param: fMinusYiUpValue: -yi*gradient of sample one
 * @param: fUpValueKernel: self dot product of sample one
 * @param: pfBlockMin: minimum value of each block (the output of this kernel)
 * @param: pnBlockMinGlobalKey: the key of each block minimum value (the output of this kernel)
 * @param: pfBlockMinYiFValue: the block minimum gradient (the output of this kernel. for convergence check)
 */
__global__ void GetBlockMinLowValue(float_point *pfYiFValue, float_point *pfAlpha, int *pnLabel, float_point fNCost,
									int nNumofTrainingSamples, float_point *pfDiagHessian, float_point *pfHessianRow,
									float_point fMinusYiUpValue, float_point fUpValueKernel, float_point *pfBlockMin,
									int *pnBlockMinGlobalKey, float_point *pfBlockMinYiFValue)
{
	__shared__ int nTempKey[BLOCK_SIZE];
	__shared__ float_point fTempMinValues[BLOCK_SIZE];

	int nThreadId = threadIdx.x;
	int nGlobalIndex = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;//global index for thread

	fTempMinValues[nThreadId] = FLT_MAX;
	//fTempMinYiFValue[nThreadId] = FLT_MAX;

	//fill data (-b_ij * b_ij/a_ij) into a block
	float_point fYiGValue;
	float_point fBeta;
	int nReduce = NOREDUCE;
	if(nGlobalIndex < nNumofTrainingSamples)
	{
		float_point fUpValue = fMinusYiUpValue;
		fYiGValue = pfYiFValue[nGlobalIndex];
		float_point fAlpha = pfAlpha[nGlobalIndex];

		int nLabel = pnLabel[nGlobalIndex];

		nTempKey[nThreadId] = nGlobalIndex;
		float_point fBUp_j;

		/*************** calculate b_ij ****************/
		//b_ij = -Gi + Gj in paper, but b_ij = -Gi + y_j * Gj in the code of libsvm. Here we follow the code of libsvm
		fBUp_j = fUpValue + fYiGValue;

	    if(((nLabel > 0) && (fAlpha > 0)) ||
	       ((nLabel < 0) && (fAlpha < fNCost))
	    	)
		{
	    	float_point fAUp_j;
			fAUp_j = fUpValueKernel + pfDiagHessian[nGlobalIndex] - 2 * pfHessianRow[nGlobalIndex];

			if(fAUp_j <= 0)
			{
				fAUp_j = TAU;
			}

		    if(fBUp_j > 0)
		    {
		    	nReduce = REDUCE1 | REDUCE0;
		    }
		    else
		    	nReduce = REDUCE0;

	    	//for getting optimized pair
			fBeta = -(fBUp_j * fBUp_j / fAUp_j);
	    	//fTempMinYiFValue[nThreadId] = -fYiGValue;
		}
	}

	if((nReduce & REDUCE0) != 0)
	{
		fTempMinValues[threadIdx.x] = -fYiGValue;
	}
	__syncthreads();
	GetMinValueOriginal(fTempMinValues, blockDim.x);
	int nBlockId;
	if(nThreadId == 0)
	{
		nBlockId =  blockIdx.y * gridDim.x + blockIdx.x;
		pfBlockMinYiFValue[nBlockId] = fTempMinValues[0];
	}

	fTempMinValues[threadIdx.x] = ((nReduce & REDUCE1) != 0) ? fBeta : FLT_MAX;

	//block level reduce
	__syncthreads();
	GetMinValueOriginal(fTempMinValues, nTempKey, blockDim.x);


	if(nThreadId == 0)
	{
		pfBlockMin[nBlockId] = fTempMinValues[0];
		pnBlockMinGlobalKey[nBlockId] = nTempKey[0];
	}
}

/*
 * @brief: kernel function for getting the minimum value in a set of block min values
 * @param: pfBlockMin: a set of min value returned from block level reducer
 * @param: pnBlockMinKey: a set of indices for block min (i.e., each block min value has a global index)
 * @param: nNumofBlock:	  the number of blocks
 * @param: pfMinValue:	  a pointer to global min value (the result of this function)
 * @param: pnMinKey:	  a pointer to the index of the global min value (the result of this function)
 */
__global__ void GetGlobalMin(float_point *pfBlockMin, int *pnBlockMinKey, int nNumofBlock,
							 float_point *pfYiFValue, float_point *pfHessianRow, float_point *pfTempKeyValue)
{
	__shared__ int nTempKey[BLOCK_SIZE];
	__shared__ float_point pfTempMin[BLOCK_SIZE];
	int nThreadId = threadIdx.x;

	if(nThreadId < nNumofBlock)
	{
		nTempKey[nThreadId] = pnBlockMinKey[nThreadId];
		pfTempMin[nThreadId] = pfBlockMin[nThreadId];
	}
	else
	{
		//nTempKey[nThreadId] = pnBlockMinKey[nThreadId];
		pfTempMin[nThreadId] = FLT_MAX;
	}
	//if the size of block is larger than the BLOCK_SIZE, we make the size to be not larger than BLOCK_SIZE
	if(nNumofBlock > BLOCK_SIZE)
	{
		float_point fTempMin = pfTempMin[nThreadId];
		int nTempMinKey = nTempKey[nThreadId];
		for(int i = nThreadId + BLOCK_SIZE; i < nNumofBlock; i += blockDim.x)
		{
			float_point fTempBlockMin = pfBlockMin[i];
			if(fTempBlockMin < fTempMin)
			{
			//store the minimum value and the corresponding key
				fTempMin = fTempBlockMin;
				nTempMinKey = pnBlockMinKey[i];
			}
		}
		nTempKey[nThreadId] = nTempMinKey;
		pfTempMin[nThreadId] = fTempMin;
	}
	 __syncthreads();	//wait until the thread within the block

	 GetMinValue(pfTempMin, nTempKey, nNumofBlock);

	 if(nThreadId == 0)
	 {
		 *(pfTempKeyValue) = (float_point)nTempKey[0];
		 if(pfYiFValue != NULL)
			 *(pfTempKeyValue + 1) = pfYiFValue[nTempKey[0]];//pfTempMin[0];
		 else
			 *(pfTempKeyValue + 1) = pfTempMin[0];

		 if(pfHessianRow != NULL)
			 *(pfTempKeyValue + 2) = pfHessianRow[nTempKey[0]];
	 }
}

/*
 * @brief: kernel function for getting the minimum value in a set of block min values
 * @param: pfBlockMin: a set of min value returned from block level reducer
 * @param: pnBlockMinKey: a set of indices for block min (i.e., each block min value has a global index)
 * @param: nNumofBlock:	  the number of blocks
 * @param: pfMinValue:	  a pointer to global min value (the result of this function)
 * @param: pnMinKey:	  a pointer to the index of the global min value (the result of this function)
 */
__global__ void GetGlobalMin(float_point *pfBlockMin, int nNumofBlock, float_point *pfTempKeyValue)
{
	__shared__ float_point pfTempMin[BLOCK_SIZE];
	int nThreadId = threadIdx.x;

	if(nThreadId < nNumofBlock)
	{
		pfTempMin[nThreadId] = pfBlockMin[nThreadId];
	}

	//if the size of block is larger than the BLOCK_SIZE, we make the size to be not larger than BLOCK_SIZE
	if(nNumofBlock > BLOCK_SIZE)
	{
		float_point fTempMin = pfTempMin[nThreadId];
		for(int i = nThreadId + BLOCK_SIZE; i < nNumofBlock; i += blockDim.x)
		{
			float_point fTempBlockMin = pfBlockMin[i];
			fTempMin = (fTempBlockMin < fTempMin) ? fTempBlockMin : fTempMin;
		}
		pfTempMin[nThreadId] = fTempMin;
	}
	 __syncthreads();	//wait until the thread within the block

	 GetMinValue(pfTempMin, nNumofBlock);

	 if(nThreadId == 0)
	 {
		*(pfTempKeyValue + 3) = pfTempMin[0];
	 }
}

/*
 * @brief: update gradient values for all samples
 * @param: pfYiFValue: the gradient of samples (input and output of this kernel)
 * @param: pfHessianRow1: the Hessian row of sample one
 * @param: pfHessianRow2: the Hessian row of sample two
 * @param: fY1AlphaDiff: the difference of old and new alpha of sample one
 * @param: fY2AlphaDiff: the difference of old and new alpha of sample two
 */
__global__ void UpdateYiFValueKernel(float_point *pfAlpha, float_point *pDevBuffer, float_point *pfYiFValue, float_point *pfHessianRow1, float_point *pfHessianRow2,
							   float_point fY1AlphaDiff, float_point fY2AlphaDiff, int nNumofTrainingSamples)
{
	if(threadIdx.x < 2)
	{
		int nTemp = int(pDevBuffer[threadIdx.x * 2]);
		pfAlpha[nTemp] = pDevBuffer[threadIdx.x * 2 + 1];
		//nTemp = int(pDevBuffer[2]);
		//pfAlpha[nTemp] = pDevBuffer[3];
	}
	__syncthreads();
	float_point fsY1AlphaDiff;
	fsY1AlphaDiff = fY1AlphaDiff;
	float_point fsY2AlphaDiff;
	fsY2AlphaDiff = fY2AlphaDiff;

	int nGlobalIndex = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;//global index for thread

	if(nGlobalIndex < nNumofTrainingSamples)
	{
		//update YiFValue
		pfYiFValue[nGlobalIndex] += (fsY1AlphaDiff * pfHessianRow1[nGlobalIndex] + fsY2AlphaDiff * pfHessianRow2[nGlobalIndex]);
	}

}
