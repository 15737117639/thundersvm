/*
 * initCuda.cpp
 *
 *  Created on: 10/12/2014
 *      Author: Zeyi Wen
 */

#include <hip/hip_runtime_api.h>
#include <iostream>

using std::cout;
using std::cerr;
using std::endl;

/**
 * @brief: initialize CUDA device
 */
bool InitCUDA()
{
    int count;

    checkCudaErrors(hipGetDeviceCount(&count));
    if(count == 0)
    {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    hipDeviceProp_t prop;
    checkCudaErrors(hipSetDevice(0));
    if(hipGetDeviceProperties(&prop, 0) == hipSuccess)
    {
        cout << "using " << prop.name << endl;
    }

	if(hipGetLastError() != hipSuccess)
	{
		cerr << "cuda error after initCuda" << endl;
		return false;
	}

    return true;
}

