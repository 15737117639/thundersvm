/*
 * initCuda.cpp
 *
 *  Created on: 10/12/2014
 *      Author: Zeyi Wen
 */

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>

using std::cout;
using std::cerr;
using std::endl;

/**
 * @brief: initialize CUDA device
 */

bool InitCUDA(hipCtx_t &context, char gpuType = 'T')
{
    int count;

    checkCudaErrors(hipGetDeviceCount(&count));
    if(count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    hipDevice_t device;
    int i;
    for(i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        checkCudaErrors(hipGetDeviceProperties(&prop, i));
        if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
        	cout << prop.name << endl;
        	if(prop.name[0] == gpuType && prop.name[1] == 'e')
        	{//prefere to use Tesla card
        		cout << "Using " << prop.name << "; device id is " << i << endl;
        		checkCudaErrors(hipSetDevice(i));
        		hipDeviceGet(&device, i);
        		hipCtxCreate(&context, hipDeviceMapHost, device);

    			hipGetDeviceProperties(&prop, i);
    			if(!prop.canMapHostMemory)
					fprintf(stderr, "Device %d cannot map host memory!\n", i);

    			break;
        	}
            if(prop.major >= 1)
            {
            	cout << "compute capability: " << prop.major << "; " << count << " devices" << endl;
            }
        }
    }

    cout << i << " v.s. " << count << endl;
    if(i == count)
    {
        hipDeviceProp_t prop;
        cout << "There is no device of \"" << gpuType << "\" series" << endl;
        checkCudaErrors(hipGetDeviceProperties(&prop, 0));

        cout << "using " << prop.name << endl;
        checkCudaErrors(hipSetDevice(0));
        hipDeviceGet(&device, i);
		hipCtxCreate(&context, hipDeviceMapHost, device);
    }

    return true;
}

