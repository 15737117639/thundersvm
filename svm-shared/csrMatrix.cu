/*
 * @author: shijiashuai
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "csrMatrix.h"

/**
 * @brief: CSR matrix constructor; construct from libsvm format data.
 */
CSRMatrix::CSRMatrix(const vector<vector<svm_node> > &samples, int numOfFeatures) : samples(samples),
                                                                                    numOfFeatures(numOfFeatures) {
    int start = 0;
    for (int i = 0; i < samples.size(); ++i) {
        csrRowPtr.push_back(start);
        int size = samples[i].size() - 1; //ignore end node for libsvm data format
        start += size;
        float_point sum = 0;
        for (int j = 0; j < size; ++j) {
            csrVal.push_back(samples[i][j].value);
            sum += samples[i][j].value * samples[i][j].value;
            csrColInd.push_back(samples[i][j].index - 1);//libsvm data format is one-based, convert it to zero-based
        }
        csrValSelfDot.push_back(sum);
    }
    csrRowPtr.push_back(start);
}

/**
 * @brief: get the number of nonzero elements of the CSR matrix.
 */
int CSRMatrix::getNnz() const {
    return csrVal.size();
}

const float_point *CSRMatrix::getCSRVal() const {
    return csrVal.data();
}

const float_point *CSRMatrix::getCSRValSelfDot() const {
    return csrValSelfDot.data();
}

const int *CSRMatrix::getCSRRowPtr() const {
    return csrRowPtr.data();
}

const int *CSRMatrix::getCSRColInd() const {
    return csrColInd.data();
}

int CSRMatrix::getNumOfSamples() const {
    return samples.size();
}

int CSRMatrix::getNumOfFeatures() const {
    return numOfFeatures;
}

/**
 * @brief: multiple two sparse matrices and output a dense matrixC.
 * @k: the dimension of training data.
 */
void CSRMatrix::CSRmm2Dense(hipsparseHandle_t handle, hipsparseOperation_t transA, hipsparseOperation_t transB, int m, int n,
                       int k, const hipsparseMatDescr_t descrA, const int nnzA, const float *valA, const int *rowPtrA,
                       const int *colIndA, const hipsparseMatDescr_t descrB, const int nnzB, const float *valB,
                       const int *rowPtrB, const int *colIndB, float *matrixC) {
    /*
     * The CSRmm2Dense result is column-major instead of row-major. To avoid transposing the result
     * we compute B'A' instead of AB' : (AB)' = B'A'
     * */
    if (transA == HIPSPARSE_OPERATION_NON_TRANSPOSE)
        transA = HIPSPARSE_OPERATION_TRANSPOSE;
    else transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    if (transB == HIPSPARSE_OPERATION_NON_TRANSPOSE)
        transB = HIPSPARSE_OPERATION_TRANSPOSE;
    else transB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    float_point *devA;
    checkCudaErrors(hipMalloc((void**)&devA,sizeof(float_point)*m*k));
    hipsparseScsr2dense(handle,m,k,descrA,valA,rowPtrA,colIndA,devA,m);
    float one(1);
    float zero(0);
    hipsparseScsrmm2(handle,transB,transA,n,m,k,nnzB,&one,descrB,valB,rowPtrB,colIndB,devA,m,&zero,matrixC,n);
    checkCudaErrors(hipFree(devA));
/**
 * the code below is csr * csr, much slower than the code above.
 */
//    if (transA == HIPSPARSE_OPERATION_NON_TRANSPOSE)
//        transA = HIPSPARSE_OPERATION_TRANSPOSE;
//    else transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
//    if (transB == HIPSPARSE_OPERATION_NON_TRANSPOSE)
//        transB = HIPSPARSE_OPERATION_TRANSPOSE;
//    else transB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
//    hipsparseMatDescr_t descrC = descrA;
//    int baseC, nnzC; // nnzTotalDevHostPtr points to host memory
//    int *nnzTotalDevHostPtr = &nnzC;
//    hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);
//    int *colIndC;
//    float *valC;
//    int *rowPtrC;
//    checkCudaErrors(hipMalloc((void **) &rowPtrC, sizeof(int) * (n + 1)));
//    hipsparseXcsrgemmNnz(handle, transB, transA, n, m, k, descrB, nnzB, rowPtrB, colIndB, descrA, nnzA, rowPtrA,
//                        colIndA, descrC, rowPtrC, nnzTotalDevHostPtr);
//    if (NULL != nnzTotalDevHostPtr) { nnzC = *nnzTotalDevHostPtr; }
//    else {
//        checkCudaErrors(hipMemcpy(&nnzC, rowPtrC + m, sizeof(int), hipMemcpyDeviceToHost));
//        checkCudaErrors(hipMemcpy(&baseC, rowPtrC, sizeof(int), hipMemcpyDeviceToHost));
//        nnzC -= baseC;
//    }
//    checkCudaErrors(hipMalloc((void **) &colIndC, sizeof(int) * nnzC));
//    checkCudaErrors(hipMalloc((void **) &valC, sizeof(float) * nnzC));
//    hipsparseScsrgemm(handle, transB, transA, n, m, k, descrB, nnzB, valB, rowPtrB, colIndB, descrA, nnzA,
//                     valA, rowPtrA, colIndA, descrC, valC, rowPtrC, colIndC);
//    hipsparseScsr2dense(handle, n, m, descrC, valC, rowPtrC, colIndC, matrixC, n);
//    checkCudaErrors(hipFree(colIndC));
//    checkCudaErrors(hipFree(valC));
//    checkCudaErrors(hipFree(rowPtrC));
}

/**
 * @brief: copy the CSR matrix to device memory.
 */
void CSRMatrix::copy2Dev(float_point *&devVal, int *&devRowPtr, int *&devColInd) {

    int nnz = this->getNnz();
    checkCudaErrors(hipMalloc((void **) &devVal, sizeof(float_point) * nnz));
    checkCudaErrors(hipMalloc((void **) &devRowPtr, sizeof(int) * (this->getNumOfSamples() + 1)));
    checkCudaErrors(hipMalloc((void **) &devColInd, sizeof(int) * nnz));
    checkCudaErrors(hipMemcpy(devVal, this->getCSRVal(), sizeof(float_point) * nnz, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(devRowPtr, this->getCSRRowPtr(), sizeof(int) * (this->getNumOfSamples() + 1),
                               hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(devColInd, this->getCSRColInd(), sizeof(int) * nnz, hipMemcpyHostToDevice));
}

/**
 * @brief: release the device CSR matrix
 */
void CSRMatrix::freeDev(float_point *&devVal, int *&devRowPtr, int *&devColInd) {
    checkCudaErrors(hipFree(devVal));
    checkCudaErrors(hipFree(devRowPtr));
    checkCudaErrors(hipFree(devColInd));
}

