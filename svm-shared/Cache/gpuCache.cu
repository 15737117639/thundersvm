#include "hip/hip_runtime.h"
//
// Created by shijiashuai on 2016/12/16.
//

#include "gpuCache.h"
#include "../constant.h"


void GpuCache::enable(int i, int j, const SvmProblem &subProblem) {
    //enable shared cache for class i and j
    this->subProblem = &subProblem;
    hessianCalculator = new DeviceHessianOnFly(subProblem, param.gamma);
    checkCudaErrors(hipMallocPitch((void **) &(devSharedCache[i]),
                                    &sizeOfEachRowInCache[i], problem.count[i] * sizeof(float_point), cacheSize[i]));
    checkCudaErrors(hipMallocPitch((void **) &(devSharedCache[j]),
                                    &sizeOfEachRowInCache[j], problem.count[j] * sizeof(float_point), cacheSize[j]));
    numOfElementEachRowInCache[i] = sizeOfEachRowInCache[i] / sizeof(float_point);
    numOfElementEachRowInCache[j] = sizeOfEachRowInCache[j] / sizeof(float_point);
    int uniqueCacheRowLength = problem.count[j];
    int uniqueCacheSize = min(CACHE_SIZE * 1024 * 1024 / 4 / uniqueCacheRowLength, cacheSize[i]);
    printf("unique cache 0 row length %d, size %d\n", uniqueCacheRowLength, uniqueCacheSize);
    checkCudaErrors(hipMallocPitch((void **) &devUniqueCache[0],
                                    &sizeOfEachRowInUniqueCache[0],
                                    uniqueCacheRowLength * sizeof(float_point),
                                    uniqueCacheSize));
    numOfElementEachRowInUniqueCache[0] = sizeOfEachRowInUniqueCache[0] / sizeof(float_point);
    uniqueCacheStrategy[0] = new CLATCache(problem.count[i]);
    uniqueCacheStrategy[0]->SetCacheSize(uniqueCacheSize);
    uniqueCacheStrategy[0]->InitializeCache(uniqueCacheSize,problem.count[i]);

    uniqueCacheRowLength = problem.count[i];
    uniqueCacheSize = min(CACHE_SIZE * 1024 * 1024 / 4 / uniqueCacheRowLength, cacheSize[j]);
    printf("unique cache 1 row length %d, size %d\n", uniqueCacheRowLength, uniqueCacheSize);
    checkCudaErrors(hipMallocPitch((void **) &devUniqueCache[1],
                                    &sizeOfEachRowInUniqueCache[1],
                                    uniqueCacheRowLength * sizeof(float_point),
                                    uniqueCacheSize));
    numOfElementEachRowInUniqueCache[1] = sizeOfEachRowInUniqueCache[1] / sizeof(float_point);
    uniqueCacheStrategy[1] = new CLATCache(problem.count[j]);
    uniqueCacheStrategy[1]->SetCacheSize(uniqueCacheSize);
    uniqueCacheStrategy[1]->InitializeCache(uniqueCacheSize,problem.count[j]);
    checkCudaErrors(hipMemcpy2D(
            devSharedCache[i], sizeOfEachRowInCache[i],
            hostSharedCache[i], problem.count[i] * sizeof(float_point),
            problem.count[i] * sizeof(float_point), cacheSize[i], hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy2D(
            devSharedCache[j], sizeOfEachRowInCache[j],
            hostSharedCache[j], problem.count[j] * sizeof(float_point),
            problem.count[j] * sizeof(float_point), cacheSize[j], hipMemcpyHostToDevice));
}

void GpuCache::disable(int i, int j) {
    delete hessianCalculator;
    delete uniqueCacheStrategy[0];
    delete uniqueCacheStrategy[1];
    checkCudaErrors(hipMemcpy2D(
            hostSharedCache[i], problem.count[i] * sizeof(float_point),
            devSharedCache[i], sizeOfEachRowInCache[i],
            problem.count[i] * sizeof(float_point), cacheSize[i], hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy2D(
            hostSharedCache[j], problem.count[j] * sizeof(float_point),
            devSharedCache[j], sizeOfEachRowInCache[j],
            problem.count[j] * sizeof(float_point), cacheSize[j], hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(devSharedCache[i]));
    checkCudaErrors(hipFree(devSharedCache[j]));
    checkCudaErrors(hipFree(devUniqueCache[0]));
    checkCudaErrors(hipFree(devUniqueCache[1]));
}

GpuCache::GpuCache(const SvmProblem &problem, const SVMParam &param) :
        problem(problem), param(param),
        numOfElementEachRowInCache(problem.getNumOfClasses()),
        devSharedCache(problem.getNumOfClasses(), NULL),
        sizeOfEachRowInCache(problem.getNumOfClasses()),
        devUniqueCache(2),
        uniqueCacheStrategy(2),
        numOfElementEachRowInUniqueCache(2),
        sizeOfEachRowInUniqueCache(2) {
    for (int i = 0; i < problem.getNumOfClasses(); ++i) {
        int rowLength = problem.count[i];
        sharedCacheStrategy.push_back(new CLATCache(rowLength));
        cacheSize.push_back(min(CACHE_SIZE * 1024 * 1024 / 4 / rowLength / 3, rowLength));
        printf("shared cache %d size=%d, #samples in class %d=%d\n", i, cacheSize[i], i, rowLength);
        sharedCacheStrategy[i]->SetCacheSize(cacheSize[i]);
        sharedCacheStrategy[i]->InitializeCache(cacheSize[i], rowLength);
        hostSharedCache.push_back(new float_point[cacheSize[i] * rowLength]);
    }
}

GpuCache::~GpuCache() {
    for (int i = 0; i < problem.getNumOfClasses(); ++i) {
        delete sharedCacheStrategy[i];
        delete[] hostSharedCache[i];
    }
}

void GpuCache::getHessianRow(int rowIndex, float_point *devHessianRow) {
    int originalLabel = subProblem->originalLabel[rowIndex];

//    printf("query row %d, label %d, ", rowIndex, originalLabel);
    //map +1 -1 to 0 1
    int label = 1 - (subProblem->v_nLabels[rowIndex] + 1) / 2;
    int sharedCacheStart = subProblem->start[label];
    int uniqueCacheStart = subProblem->start[1 - label];
    int sharedCacheCount = subProblem->count[label];
    int uniqueCacheCount = subProblem->count[1 - label];
//    printf("original label %d, label %d\n",originalLabel, label);
//    printf("shared cache start %d, unique cache start %d\n",sharedCacheStart,uniqueCacheStart);
//    printf("row index %d\n",rowIndex);


    int cacheLocation;
    bool cacheFull = false;

    //query unique cache
    int uniqueCacheOffset = -subProblem->start[label];
    bool cacheHit = uniqueCacheStrategy[label]->GetDataFromCache(rowIndex + uniqueCacheOffset, cacheLocation, cacheFull);
    if (!cacheHit) {
        if (cacheFull)
            uniqueCacheStrategy[label]->ReplaceExpired(rowIndex + uniqueCacheOffset, cacheLocation, NULL);
//        printf("unique cache miss, save to location %d, ", cacheLocation);
        hessianCalculator->ReadRow(rowIndex,
//                                   devHessianRow+uniqueCacheStart,
                                   devUniqueCache[label] + cacheLocation * numOfElementEachRowInUniqueCache[label],
                                   uniqueCacheStart,
                                   uniqueCacheStart + uniqueCacheCount);
    } else {
//        printf("unique cache hit at %d, ", cacheLocation);
    };
    checkCudaErrors(hipMemcpy(
            devHessianRow + uniqueCacheStart,
            devUniqueCache[label] + cacheLocation * numOfElementEachRowInUniqueCache[label],
            sizeof(float_point) * uniqueCacheCount,
            hipMemcpyDeviceToDevice));

    //query shared cache
    int sharedCacheOffset = -subProblem->start[label];
//    printf("offset is %d, ", sharedCacheOffset);
    cacheHit = sharedCacheStrategy[originalLabel]->GetDataFromCache(rowIndex + sharedCacheOffset, cacheLocation,
                                                                    cacheFull);
    if (!cacheHit) {
        if (cacheFull)
            sharedCacheStrategy[originalLabel]->ReplaceExpired(rowIndex + sharedCacheOffset, cacheLocation, NULL);
//        printf("shared cache %d miss, save to location %d.\n", originalLabel, cacheLocation);
        hessianCalculator->ReadRow(rowIndex,
                                   devSharedCache[originalLabel] +
                                   cacheLocation * numOfElementEachRowInCache[originalLabel],
//                                   devHessianRow+sharedCacheStart,
                                   sharedCacheStart,
                                   sharedCacheStart + sharedCacheCount);
    } else {
//        printf("shared cache %d hit at %d.\n", originalLabel, cacheLocation);
    }
    checkCudaErrors(hipMemcpy(
            devHessianRow + sharedCacheStart,
            devSharedCache[originalLabel] + cacheLocation * numOfElementEachRowInCache[originalLabel],
            sizeof(float_point) * sharedCacheCount,
            hipMemcpyDeviceToDevice));
}
