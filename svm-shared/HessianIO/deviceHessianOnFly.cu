#include "hip/hip_runtime.h"
//
// Created by ss on 16-11-15.
//

#include "deviceHessianOnFly.h"
#include "../constant.h"

__global__ void RBFKernel(const float_point *aSelfDot, float_point bSelfDot, float_point *dotProduct, int numOfSamples,
                          float gamma) {
    const int idx = blockDim.x * blockIdx.x + threadIdx.x;
//    __shared__ float_point _bSelfDot;
//    if (0 == idx)
//        _bSelfDot = bSelfDot;
//    __syncthreads();
    if (idx < numOfSamples) {
        dotProduct[idx] = expf(-(aSelfDot[idx] + bSelfDot - dotProduct[idx] * 2) * gamma);
    }
}

void DeviceHessianOnFly::ReadRow(int nPosofRowAtHessian, float_point *devHessianRow, int start, int end) {
//    end = csrMat.getNumOfSamples();
//    printf("start %d, end %d\n",start,end);
    const int numOfSamples = end - start;
    const int *csrRowPtr = csrMat.getCSRRowPtr();
    const int numOfFeatures = csrMat.getNumOfFeatures();
    const int nnzA = csrRowPtr[end] - csrRowPtr[start];
    const int *devARowPtr = devRowPtrSplit + start;
    if (start!=0)
        devARowPtr++;
    const float_point *devAVal = devVal + csrRowPtr[start];
    const int *devAColInd = devColInd + csrRowPtr[start];
    const int nnzB = csrRowPtr[nPosofRowAtHessian + 1] - csrRowPtr[nPosofRowAtHessian];
    const float_point *devBVal = devVal + csrRowPtr[nPosofRowAtHessian];
    const int *devBColInd = devColInd + csrRowPtr[nPosofRowAtHessian];
    float_point *devBDense;
    checkCudaErrors(hipMalloc((void **) &devBDense, sizeof(float_point) * numOfFeatures));
    checkCudaErrors(hipMemset(devBDense, 0, sizeof(float_point) * numOfFeatures));
    hipsparseSsctr(handle, nnzB, devBVal, devBColInd, devBDense, HIPSPARSE_INDEX_BASE_ZERO);
    checkCudaErrors(hipMemset(devHessianRow,0,sizeof(float_point) * numOfSamples));
//    if (numOfSamples != 100) {
        hipsparseScsrmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                       numOfSamples, 1, numOfFeatures,
                       nnzA, &one, descr, devAVal, devARowPtr, devAColInd,
                       devBDense, numOfFeatures, &zero,
                       devHessianRow, numOfSamples);
//    }
//    else {
//        hipsparseScsrmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
//                       numOfSamples, 1, numOfFeatures,
//                       nnzA, &one, descr, devVal, devRowPtr, devColInd,
//                       devBDense, numOfFeatures, &zero,
//                       devHessianRow, numOfSamples);
//    }
    RBFKernel << < Ceil(numOfSamples, BLOCK_SIZE), BLOCK_SIZE >> >
            (devValSelfDot + start, csrMat.csrValSelfDot[nPosofRowAtHessian], devHessianRow, numOfSamples, gamma);
//    float_point *hrow = new float_point[numOfSamples];
//    checkCudaErrors(
//            hipMemcpy(hrow, devHessianRow, sizeof(float_point) * numOfSamples, hipMemcpyDeviceToHost));
//    RBFKernelFunction function(gamma);
//    float_point *hostKernel = new float_point[problem.getNumOfSamples()];
//    float_point totalErr = 0;
//    vector<vector<svm_node> > s = problem.v_vSamples;
//    function.ComputeSparseRow(s,nPosofRowAtHessian,1,hostKernel);
//    for (int i = 0; i < problem.getNumOfSamples(); ++i) {
//       float_point err = fabs(hostKernel[i] - hrow[i]);
//        totalErr +=err;
//        printf("row %d, col %d, host %f, device %f,err %f\n",nPosofRowAtHessian, i, hostKernel[i],hrow[i],err);
//    }
//    printf("compute row %d, total err %f\n",nPosofRowAtHessian,totalErr);
//    memcpy(devHessianRow,hostKernel,sizeof(float_point) * numOfSamples);
//    delete[] hostKernel;
    checkCudaErrors(hipFree(devBDense));
//    checkCudaErrors(hipFree(devC));
}

bool DeviceHessianOnFly::PrecomputeHessian(const string &strHessianMatrixFileName, const string &strDiagHessianFileName,
                                         vector<vector<float_point> > &v_v_DocVector) {
    return true;
}

bool DeviceHessianOnFly::GetHessianDiag(const string &strFileName, const int &nNumofTraingSamples,
                                      float_point *pfHessianDiag) {
    for (int i = 0; i < nNumofTraingSamples; ++i) {
        pfHessianDiag[i] = 1;
    }
    return true;
}

DeviceHessianOnFly:: DeviceHessianOnFly(const SvmProblem &subProblem, float_point gamma) :
        gamma(gamma), zero(0.0f), one(1.0f),
        csrMat(subProblem.v_vSamples, subProblem.getNumOfFeatures()) {
    hipsparseCreate(&handle);
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    checkCudaErrors(hipMalloc((void **) &devVal, sizeof(float_point) * csrMat.getNnz()));
    checkCudaErrors(hipMalloc((void **) &devValSelfDot, sizeof(float_point) * csrMat.getNumOfSamples()));
    checkCudaErrors(hipMalloc((void **) &devRowPtr, sizeof(int) * (csrMat.getNumOfSamples() + 1)));
    checkCudaErrors(hipMalloc((void **) &devRowPtrSplit, sizeof(int) * (csrMat.getNumOfSamples() + 2)));
    checkCudaErrors(hipMalloc((void **) &devColInd, sizeof(int) * (csrMat.getNnz())));
    checkCudaErrors(hipMemcpy(devVal, csrMat.getCSRVal(), sizeof(float_point) * csrMat.getNnz(),
                               hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(devValSelfDot, csrMat.getCSRValSelfDot(),
                               sizeof(float_point) * subProblem.v_vSamples.size(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(devRowPtr, csrMat.getCSRRowPtr(), sizeof(int) * (subProblem.v_vSamples.size() + 1),
                               hipMemcpyHostToDevice));
    //nnz for samples with label +1
    int nnzA = csrMat.csrRowPtr[subProblem.count[0]];
    csrRowPtrSplit = vector<int>(csrMat.csrRowPtr.begin(),csrMat.csrRowPtr.begin()+subProblem.count[0]+1);
    for (int i = 0; i <= subProblem.count[1]; ++i) {
        csrRowPtrSplit.push_back(csrMat.csrRowPtr[subProblem.count[0] + i] - nnzA);
    }
    checkCudaErrors(hipMemcpy(devRowPtrSplit, csrRowPtrSplit.data(), sizeof(int) * (subProblem.v_vSamples.size() + 2),
                               hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(devColInd, csrMat.getCSRColInd(), sizeof(int) * (csrMat.getNnz()),
                               hipMemcpyHostToDevice));

}

//bool HostHessianOnFly::AllocateBuffer(int nNumofRows) {
//    checkCudaErrors(hipHostMalloc((void**)&m_pfHessianRows,sizeof(float_point)*m_nTotalNumofInstance*nNumofRows));
////    return BaseHessian::AllocateBuffer(nNumofRows);
//    return true;
//}
//
//bool HostHessianOnFly::ReleaseBuffer() {
////    return BaseHessian::ReleaseBuffer();
//    checkCudaErrors(hipHostFree(m_pfHessianRows));
//    return true;
//}

