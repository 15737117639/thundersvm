#include "hip/hip_runtime.h"
//
// Created by ss on 16-11-15.
//

#include "deviceHessianOnFly.h"
#include "../constant.h"

__global__ void RBFKernel(const float_point *aSelfDot, int bRow, float_point *dotProduct, int numOfSamples,
                          float gamma) {
    const int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < numOfSamples) {
        const float bSelfDot = aSelfDot[bRow];
        dotProduct[idx] = expf(-(aSelfDot[idx] + bSelfDot - dotProduct[idx] * 2) * gamma);
    }
}

void DeviceHessianOnFly::ReadRow(int nPosofRowAtHessian, float_point *pfHessianRow) {
//    printf("READ ROW %d\n", nPosofRowAtHessian);
    const int nnzB = problem.getCSRRowPtr()[nPosofRowAtHessian + 1] - problem.getCSRRowPtr()[nPosofRowAtHessian];
    const float_point *devBVal = devValA + problem.getCSRRowPtr()[nPosofRowAtHessian];
    const int *devBInd = devColIndA + problem.getCSRRowPtr()[nPosofRowAtHessian];
    float_point *devBDense;
    checkCudaErrors(hipMalloc((void **) &devBDense, sizeof(float_point) * problem.getNumOfFeatures()));
    checkCudaErrors(hipMemset(devBDense, 0, sizeof(float_point) * problem.getNumOfFeatures()));
    hipsparseSsctr(handle, nnzB, devBVal, devBInd, devBDense, HIPSPARSE_INDEX_BASE_ZERO);
    float_point *devC;
    checkCudaErrors(hipMalloc((void **) &devC, sizeof(float_point) * problem.getNumOfSamples()));
    checkCudaErrors(hipMemset(devC,0,sizeof(float_point) * problem.getNumOfSamples()));
    hipsparseScsrmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                   problem.getNumOfSamples(), 1, problem.getNumOfFeatures(),
                   nnz, &one, descr, devValA, devRowPtrA, devColIndA,
                   devBDense, problem.getNumOfFeatures(), &zero,
                   devC, problem.getNumOfSamples());
    RBFKernel << < Ceil(problem.getNumOfSamples(), BLOCK_SIZE), BLOCK_SIZE >> >
                                                                (devValASelfDot, nPosofRowAtHessian, devC, problem.getNumOfSamples(), gamma);
    checkCudaErrors(
            hipMemcpy(pfHessianRow, devC, sizeof(float_point) * problem.getNumOfSamples(), hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(devBDense));
    checkCudaErrors(hipFree(devC));
}

bool DeviceHessianOnFly::PrecomputeHessian(const string &strHessianMatrixFileName, const string &strDiagHessianFileName,
                                         vector<vector<float_point> > &v_v_DocVector) {
    return true;
}

bool DeviceHessianOnFly::GetHessianDiag(const string &strFileName, const int &nNumofTraingSamples,
                                      float_point *pfHessianDiag) {
    for (int i = 0; i < nNumofTraingSamples; ++i) {
        pfHessianDiag[i] = 1;
    }
    return true;
}

//bool HostHessianOnFly::AllocateBuffer(int nNumofRows) {
//    checkCudaErrors(hipHostMalloc((void**)&m_pfHessianRows,sizeof(float_point)*m_nTotalNumofInstance*nNumofRows));
////    return BaseHessian::AllocateBuffer(nNumofRows);
//    return true;
//}
//
//bool HostHessianOnFly::ReleaseBuffer() {
////    return BaseHessian::ReleaseBuffer();
//    checkCudaErrors(hipHostFree(m_pfHessianRows));
//    return true;
//}

