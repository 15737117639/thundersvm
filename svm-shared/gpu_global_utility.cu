/*
 * utility.cpp
 *
 *  Created on: 16/03/2013
 *      Author: zeyi
 */
#include "gpu_global_utility.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>

using std::cout;
using std::endl;

float_point gfPCost = 4;	//cost for positive samples in training SVM model (i.e., error tolerance)
float_point gfNCost = 4;	//cost for negative samples in training SVM model
float_point gfGamma = 1;
int gNTest = 0;
